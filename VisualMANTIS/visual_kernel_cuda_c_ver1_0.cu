#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// 			     //////////////////////////////////////////////////////////
//  			     //							     //
// 			     //   	        hybridMANTIS v1.0		     //
// 			     //   	  fastDETECT2 kernel - CUDA + C  	     //
//			     //		   (optical photons transport)		     //
//			     //							     //
//			     //////////////////////////////////////////////////////////
//
// 
//
//
// ****Disclaimer****
//  This software and documentation (the "Software") were developed at the Food and Drug Administration (FDA) by employees of the Federal Government in
//  the course of their official duties. Pursuant to Title 17, Section 105 of the United States Code, this work is not subject to copyright protection
//  and is in the public domain. Permission is hereby granted, free of charge, to any person obtaining a copy of the Software, to deal in the Software
//  without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, or sell copies of the
//  Software or derivatives, and to permit persons to whom the Software is furnished to do so. FDA assumes no responsibility whatsoever for use by other
//  parties of the Software, its source code, documentation or compiled executables, and makes no guarantees, expressed or implied, about its quality,
//  reliability, or any other characteristic. Further, use of this code in no way implies endorsement by the FDA or confers any advantage in regulatory
//  decisions. Although this software can be redistributed and/or modified freely, we ask that any derivative works bear some notice that they are
//  derived from it, and any modified versions bear some notice that they have been modified. 
//
//
//	Associated publication: Sharma Diksha, Badal Andreu and Badano Aldo, "hybridMANTIS: a CPU-GPU Monte Carlo method for modeling indirect x-ray detectors with
//				columnar scintillators". Physics in Medicine and Biology, 57(8), pp. 2357–2372 (2012)
//
//
//	File:   	kernel_cuda_c_ver1_0.cu 			
//	Author: 	Diksha Sharma (US Food and Drug Administration)
//	Email: 		diksha.sharma@fda.hhs.gov			
//	Last updated:  	Apr 13, 2012
//
//	Modified Name:	visual_kernel_cuda_c_ver1_0.cu
//	Updated: 	4/23/2013
//	Author:		Han Dong (US Food and Drug Administration / University of Maryland Baltimore County
//	Email:		han6@umbc.edu
//	Comments:	This code contains modified CUDA kernels in order to retrieve photon information during execution of hybridMANTIS.
//			The algorithm and code is unchanged and thus is the same as the base hybridMANTIS code, however what was added was
//			additional data structures to save photon data during the execution so that it can be used by the visualization for
//			rendering.
//
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////
//
//      Header libraries
//
/////////////////////////////////////////

	#include <math.h>
	#include <stdio.h>
	#include <stdlib.h>
	#include <string.h>
	#include <sys/time.h>
	#include <time.h>

/////////////////////////////////////////
//
//       Constants
//
/////////////////////////////////////////

	#define twopipen 6.283185308	// 2*PI
	#define pi 3.14159265		// PI
	#define epsilon 8.1929093e-6	// a very small number for float comparisons


/////////////////////////////////////////////////////////////////////////////////////
//
//     Data structure for storing a scintillation event location and deposited energy
//
/////////////////////////////////////////////////////////////////////////////////////

	struct start_info
	{
		double str_x;		// x-coordinate
		double str_y;		// y-coordinate
		double str_z;		// z-coordinate
		double str_E;		// deposited energy
		int str_histnum;	// x-ray history #
		int str_N;		// # of optical photons to be transported for this energy
	};


//////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//	Fortran structure declarations - using PENELOPE 2006 (coded in Fortran)
//	A 'common' block in Fortran needs to be declared here to allow calling function interexchangebly.	
// 	
//////////////////////////////////////////////////////////////////////////////////////////////////////////

// Similar structure to 'start_info' - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			double xbufopt[gpubufsize];	// x-coordinate array of scintillation events
			double ybufopt[gpubufsize];	// y-coordinate
			double zbufopt[gpubufsize];	// z-coordinate
			double debufopt[gpubufsize];	// deposited energy
			int nbufopt[gpubufsize];	// x-ray history #
			int myctropt;			// equal to 'gpubufsize' (buffer size when calling fastDETECT2 in the GPU)
		        int cpu_num_real;		// equal to number of x-ray histories to be run by cpuoptical
		} optical_;
	#else
		extern struct
		{
			double xbufopt[mybufsize];
			double ybufopt[mybufsize];
			double zbufopt[mybufsize];
			double debufopt[mybufsize];	
			int nbufopt[mybufsize];		
			int myctropt;			// equal to 'mybufsize'	(buffer size when calling fastDETECT2 in the CPU)
		        int cpu_num_real;
		} optical_;
	#endif

// Storing optical output statistics - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			double glgen;			// total # optical photons generated
			double gldetect;		// total # optical photons detected
			double glabstop;		// total # optical photons absorbed at top surface (H/2)
			double glabsbulk;		// total # optical photons absorbed in the bulk
			double gllost;			// total # optical photons lost at detector boundaries
			double gloutofcol;		// total # optical photons killed when they went out of the current column (due to precision errors)
			double gltheta1;		// total # optical photons killed when theta1 > 90degrees; 100x resampling
			double glgputime;		// total GPU time (ms)
		} optstats_;
	#else
		extern struct
		{
			double glgen;
			double gldetect;
			double glabstop;
			double glabsbulk;
			double gllost;
			double gloutofcol;
			double gltheta1;
			double glgputime;		// total CPU time (ms)
		} optstats_;
	#endif

// Storing deposited energy and # optical photons detected - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			int gldetprimary[1000];		// total # optical photons detected per primary
		} outputdetprim_;
	#else
		extern struct
		{
			int gldetprimary[1000];
		} outputdetprim_;
	#endif

// Structure for storing point response functions - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			unsigned long long int newimageopt[501][501];	// storing point response function
			unsigned long long int tempimageopt[501][501];	// used in timing CPU optical
		} outputimage_;
	#else
		extern struct
		{
			unsigned long long int newimageopt[501][501];
			unsigned long long int tempimageopt[501][501];
		} outputimage_;
	#endif

// Storing the memory addresses of arrays, in order to call fastDETECT2 in the GPU asynchronously - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			unsigned long long int gpuimage;		// storing memory address of gpu image array (2D)
			unsigned long long int gpudetect;		// storing memory address of gpu detected array
			unsigned long long int hosta;			// storing memory address of host memory for x,y,z,E
			unsigned long long int deva;			// storing memory address of device memory for x,y,z,E
			unsigned long long int devpitch;		// storing memory address pitch of gpu image array (2D)
		} gpumemaddr_;
	#endif

// Storing the input arguments - declared in PENELOPE
	#ifdef USING_CUDA
		extern "C" struct
		{
			double detx;		// detector length in x (in um). x in (0,detx).
			double dety;		// detector length in y (in um). y in (0,dety).
			double detheight;	// height of a column or thickness of detector (in um). z in range (-H/2, H/2).
			double detradius;	// radius of a column (in um). Assuming same properties for all the columns.
			double detnC;		// refractive index of columns.
			double detnIC;		// refractive index of intercolumnar material.
			double dettop;		// column's top surface absorption fraction (0,1).
			double detbulk;		// column's bulk absorption coefficient (in um^-1). 
			double detbeta;		// roughness coefficient of column surface walls (0,0.5).
			double detdmin;		// minimum distance a photon can travel when transmitted from a column: on-the-fly geometry.
			double detdmax;		// maximum distance a photon can travel when transmitted from a column: on-the-fly geometry.
			double detlboundx;	// x lower bound of point response function (in um).
			double detlboundy;	// y lower bound (in um).
			double detuboundx;	// x upper bound (in um). 
			double detuboundy;	// y upper bound (in um). 
			double detyield;	// light yield (/eV).
			double detsensorRefl;	// Non-Ideal sensor reflectivity (%) (0,100).
			int detpixel;		// 1 pixel = 'pixelsize' microns (in um) - for storing PRF.
			int rungpu;		// flag to run on the GPU (value=1).
			int machinenum;		// machine number where code executes. This number is useful in differentiating output file names when using same input arguments.
			int mynumhist;		// total number of primaries to be simulated
			int minphotons;		// minimum number of optical photons detected to be included in PHS
			int maxphotons;		// maximum number of optical photons detected to be included in PHS
			int mynumbins;		// number of bins for genrating PHS
		} inputargs_;
	#else
		extern struct
		{
			double detx;
			double dety;
			double detheight;
			double detradius;
			double detnC;
			double detnIC;
			double dettop;
			double detbulk;
			double detbeta;
			double detdmin;
			double detdmax;
			double detlboundx;
			double detlboundy;
			double detuboundx;
			double detuboundy;
			double detyield;
			double detsensorRefl;
			int detpixel;
			int rungpu;
			int machinenum;
			int mynumhist;
			int minphotons;		
			int maxphotons;		
			int mynumbins;		
		} inputargs_;
	#endif

/////////////////////////////////////////////////////////////////////////////////////
//
//     Data structure for storing photon histories
//
/////////////////////////////////////////////////////////////////////////////////////
struct histStruct
{
	float x[1000];			// x-coordinate
	float y[1000];			// y-coordinate
	float z[1000];			// z-coordinate
	float Xc[1000];			// x coordinate of cylinder
	float Yc[1000];			// y coordinate of cylinder
	int terminated[1000];	//counter to indicate type of termination
	int histCounter;	// counter to keep track of number of histories
};
	
/////////////////////////////////////////
//
//       Function declarations
//
/////////////////////////////////////////

// transports optical photon from its generation until it ends (detected/absorbed/lost).
	#ifdef USING_CUDA
		__global__ void algo(struct start_info *info, unsigned long long int *myimage, int *num_detected_primary, size_t pitch, int rowsread, float xdetector, float ydetector, 
		float R, float H, float n1, float n2, float top_absfrac, float bulk_abscoeff, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, 
		float ubound_y, float d_max, float sensorRefl, struct histStruct * dPhotonHist, int boolToCollectData);	
	#else
		int algo(float *normal, float *old_pos, float *pos, float *dcos, unsigned long long int *num_rebound, int* seed, struct start_info info, 
		unsigned long long int *myimage, float xdetector, float ydetector, float R, float H, float n1, float n2, float top_absfrac, float bulk_abscoeff, float beta, 
		float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, float sensorRefl, float d_max, int ydim, int *h_num_detected_prim);
	#endif

// photon within a column. calculate if it gets absorbed or moves inside the column.
	#ifdef USING_CUDA
		__device__ inline int isotropic(float3 *pos, float3 *dcos, int2* seed, float bulk_abscoeff, float R, float H, float xdetector, float ydetector, 
		struct start_info *info, unsigned long long int mynum_rebound, float *Xc, float *Yc, int mytid, struct histStruct * dPhotonHist, int boolToCollectData);
	#else
		int isotropic(float *pos, float *dcos, int* seed, float bulk_abscoeff, float R, float H, float xdetector, float ydetector, struct start_info info, 
		unsigned long long int mynum_rebound);
	#endif

// photon within a column. calculate distance to next position in the same column and move it.
	#ifdef USING_CUDA
		__device__ float dist_to_surface(float3 *pos, float3 *dcos, float R, float H, float xdetector, float ydetector, struct start_info *info, 
		unsigned long long int mynum_rebound, float *Xc, float *Yc, int mytid, struct histStruct * dPhotonHist, int boolToCollectData);	
	#else
		float dist_to_surface(float *pos, float *dcos, float R, float H, float xdetector, float ydetector, struct start_info info, unsigned long long int mynum_rebound);
	#endif

// photon within/between columns. calculate if it gets reflected or transmitted.
	#ifdef USING_CUDA
		__device__ int boundary_analysis(float3 *normal, float3 *pos, float3 *dcos, int2* seed, float xdetector, float ydetector, float R, float H, float n1, float n2, 
		float top_absfrac, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, 
		float *Xc, float *Yc, size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float d_max, float sensorRefl, struct histStruct * dPhotonHist,
		int boolToCollectData);	
	#else
		int boundary_analysis(float *normal, float *pos, float *dcos, int* seed, float xdetector, float ydetector, float R, float H, float n1, float n2, float top_absfrac, 
		float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, 
		float d_max, float sensorRefl, int ydim, int *h_num_detected_prim);	
	#endif

// transmit photon to another column. calculates the new position when it transmits, build new column and move photon here.
	#ifdef USING_CUDA
		__device__ int transmit(float3 *pos, float3 *dcos, float3 *normal, int2* seed, float xdetector, float ydetector, float H, float top_absfrac, float beta, float d_min, int pixelsize, 
		float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, size_t pitch, struct start_info *info, int mytid, 
		int *num_detected_primary, float d_max, float sensorRefl, int flagCCT, float *Xc, float *Yc, struct histStruct * dPhotonHist, int boolToCollectData);
	#else
		int transmit(float *pos, float *dcos, float *normal, int* seed, float xdetector, float ydetector, float H, float top_absfrac, float beta, float d_min, int pixelsize, float lbound_x,
		float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, float d_max, float sensorRefl, int ydim, 
		int flagCCT, int *h_num_detected_prim);	
	#endif

// called when photon reflects from sensor plane (bottom surface) of the detector, outside of any column.
	#ifdef USING_CUDA
		__device__ int refl_bottom(float3 *pos, float3 *dcos, float3 *normal, float xdetector, float ydetector, int2* seed, 
		float beta, float d_min, float H, float d_max, int mytid, float *Xc, float *Yc, struct histStruct * dPhotonHist, int boolToCollectData);
	#else
		int refl_bottom(float *pos, float *dcos, float *normal, float xdetector, float ydetector, int* seed, float beta, float d_min, float H, float d_max);	
	#endif

// calculate dot product of two vectors to give cosine of angle between them.
	#ifdef USING_CUDA
		__device__ inline float dot_product(float3 *aa, float3 *b);	
	#else
		float dot_product(float *aa, float *b);	
	#endif

// determine if photon got detected at sensor plane or is reflected back within the column
	#ifdef USING_CUDA
		__device__ inline int detection(float3 *pos, float H, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float sensorRefl, float d_min, int2* seed, float3 *dcos, float3 *normal, 
		float bulk_abscoeff, float R, float xdetector, float ydetector, unsigned long long int mynum_rebound, float *Xc, float *Yc, struct histStruct * dPhotonHist,
		int boolToCollectData);
	#else
		int detection(float *pos, float H, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, 
		struct start_info info, float sensorRefl, float d_min, int* seed, float *dcos, float *normal, float bulk_abscoeff, float R, float xdetector, float ydetector, 
		unsigned long long int mynum_rebound, int ydim, int *h_num_detected_prim); 
	#endif

// calculate directional cosines of reflected/refracted vector.
	#ifdef USING_CUDA
		__device__ inline void trans_dir_cos(float3 *dcos, float3 *normal, float refl_theta, float trans_theta, int flag_ref, int mytid, struct start_info *info);  
	#else
		void trans_dir_cos(float *dcos, float *normal, float refl_theta, float trans_theta, int flag_ref, struct start_info info);
	#endif

// calculate new rough normal vector depending on value of 'beta' (roughness coefficient).
	#ifdef USING_CUDA
		__device__ inline void RoughSurface(float3 *normal, int2* seed, float beta);  
	#else
		void RoughSurface(float *normal, int* seed, float beta); 
	#endif



// RANECU pseudo random number generator
	#ifdef USING_CUDA
		__device__ inline void init_PRNG(int history_batch, int histories_per_thread, int seed_input, int2* seed);	// initialize the generator
		__device__ inline int abMODm(int m, int a, int s);								// calculate a1*a2 MOD m
		__device__ inline float ranecu(int2* seed);									// Pseudo RNG returning float value (single-precision)
	#else
		void init_PRNG(int history_batch, int histories_per_thread, int seed_input, int* seed);
		int abMODm(int m, int a, int s);
		float ranecu(int* seed);
	#endif


/////////////////////////////////////////
//
//       Global variables
//
/////////////////////////////////////////

	#ifdef USING_CUDA
		// counters
		__device__ unsigned long long int num_generated; // total # of photons generated for all the x-ray histories (across all threads)
		__device__ unsigned long long int num_detect;	 // total # of photons detected at the sensor plane of a column
		__device__ unsigned long long int num_abs_top;	 // total # of photons absorbed at the top surface of the detector (using 'top_absfrac')
		__device__ unsigned long long int num_abs_bulk;	 // total # of photons absorbed in the bulk of the detector (using 'bulk_abscoeff')
		__device__ unsigned long long int num_lost;	 // total # of photons lost when exiting out of the detector boundaries in x/y direction
		__device__ unsigned long long int num_outofcol;	 // total # of photons killed because they moved out of current column when reflected (due to precision errors)
		__device__ unsigned long long int num_theta1;	 // total # of photons killed if incidence angle > 1.57 or < 0 radian (after resampling max 100 times)	
		__device__ float photon_distance;     		 // total distance travelled by all the photons
		__device__ int dev_numPhotonHist;
	#else
		// counters
		unsigned long long int num_generated=0;	
		unsigned long long int num_detect=0;	
		unsigned long long int num_abs_top=0;	
		unsigned long long int num_abs_bulk=0;	
		unsigned long long int num_lost=0;	
		unsigned long long int num_outofcol=0;	 
		unsigned long long int local_counter=0;	 	// total number of photons terminated (either detected at sensor, absorbed at the top or in the bulk)
		unsigned long long int num_theta1=0;	

		//flags
		int absorbed=0;					// flag for photons absorbed at the top surface of the detector
		int detect=0;					// flag for photons detected at the sensor plane of the detector
		int bulk_abs=0;					// flag for photons absorbed in the bulk of a column

		float Xc=0.0f;					// center coordinates (x,y) of the current cylinder
		float Yc=0.0f;
		float photon_distance=0.0f;     		// total distance travelled by all the photons

		FILE *fp1;
	#endif

/////////////////////////////////////////
//
//    Functions definition
//
/////////////////////////////////////////

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// transports optical photon from its generation until it ends (detected/absorbed/lost).
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
__global__ void algo(struct start_info *info, unsigned long long int *myimage, int *num_detected_primary, size_t pitch, int rowsread, float xdetector, float ydetector,  
float R, float H, float n1, float n2, float top_absfrac, float bulk_abscoeff, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x,  
float ubound_y, float d_max, float sensorRefl, struct histStruct * dPhotonHist, int boolToCollectData)
{
	unsigned long long int local_counter = 0; 	// total # optical photons terminated (either detected at sensor (bottom surface), absorbed at top or in the bulk). Used for checking if the total number of photons has been transported for this thread.
	unsigned long long int local_num_generated = 0;	// total # optical photons generated for this thread
	unsigned long long int num_rebound=0;
	float3 dcos, normal, pos; 			// directional cosine, normal and current position vector
	float rr=0.0f, theta=0.0f;
	float Xc=0.0f;					// center x,y coordinates of the current cylinder
	float Yc=0.0f;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;	// cuda thread ID

	// flags
	int absorbed=0;			// flag for photons absorbed at the top surface of the detector (yes=1; no=0)
	int detect=0;			// flag for photons detected at the sensor plane of the detector
	int bulk_abs=0;			// flag for photons absorbed in the bulk of a column
	
	if(tid < rowsread)	// # threads launched is always > # scintillation events. Therefore, only the threads corresponding to an event works, rest do nothing. 
	{
		int NUM_EACH_THREAD = info[tid].str_N;		// number of photons to be simulated by each thread. info[tid] is the scintillation event for this thread.

		// Initialize vectors
		dcos.x = 0.0f; dcos.y = 0.0f; dcos.z = 0.0f;
		normal.x = 0.0f; normal.y = 0.0f; normal.z = 0.0f;
		pos.x = info[tid].str_x; 
		pos.y = info[tid].str_y; 
		pos.z = info[tid].str_z;	// starting location for this scintillation event.
		
		/*if(tid == 6 || tid == 7)
		{
				printf("algo - tid: %d, initial starting xyz %f %f %f\n", tid, pos.x, pos.y, pos.z);
		}*/
		
		int seed_input = 271828182+tid; 		// RANECU rng seed input
		int2 seed;

		// Initialize the RANECU generator in a position far away from the previous history
		init_PRNG(tid, 50000, seed_input, &seed);     

		// Initalize the device memory for directional cosine
		dcos.z = (ranecu(&seed) * 2.0f) - 1.0f;	// generate random number between -1.0 and 1.0

		rr = sqrt(1.0f - dcos.z*dcos.z);
		theta = ranecu(&seed) * twopipen;	// generate random number between 0 and 2pi
	
		dcos.x = rr*cos(theta);
		dcos.y = rr*sin(theta);

		// normalize
		if (((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) < (1.0f - epsilon)) || ((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) > (1.0f + epsilon)))
	 	{
			dcos.x = dcos.x/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
			dcos.y = dcos.y/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
			dcos.z = dcos.z/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
		}

		local_num_generated++;		// increment number of generated photons

		/*if(tid == 0 || tid == 1 || tid == 3 || tid == 6 || tid == 7)
		{
			printf("tid: %d before while, local_num_generated: %lld, NUM_EACH_THREAD: %d\n", tid, local_num_generated, NUM_EACH_THREAD);
		}*/
				
		while(local_num_generated < (NUM_EACH_THREAD+1))	//run until NUM_EACH_THREAD particles generated
		{	
			/*if(tid == 6 || tid == 7)
			{
				printf("tid: %d after while(local_num_generated <...\n", tid);
			}*/
		
			if(absorbed == 0)  	  // not absorbed at the top surface of detector, check for absorption in the bulk and detection    
			{
				/*if(tid == 6 || tid == 7)
				{
					printf("tid: %d calling isotropic\n", tid);
				}*/
				
				bulk_abs = isotropic(&pos, &dcos, &seed, bulk_abscoeff, R, H, xdetector, ydetector, &info[tid], num_rebound, &Xc, &Yc, tid, dPhotonHist, boolToCollectData);
				if(bulk_abs == 0)	// not absorbed in the bulk, check for detection
				{
					/*if(tid == 6 || tid == 7)
					{
						printf("tid: %d calling detection\n", tid);
					}*/
				
					detect = detection(&pos, H, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, pitch, info, tid, num_detected_primary, 
					sensorRefl, d_min, &seed, &dcos, &normal, bulk_abscoeff, R, xdetector, ydetector, num_rebound, &Xc, &Yc, dPhotonHist, boolToCollectData);
				}
			}
		 
			if( ((detect == 1) || (absorbed == 1) || (bulk_abs == 1)) && (local_counter < (NUM_EACH_THREAD-1)) ) // particle terminated
			{
				/*if(tid == 6 || tid == 7)
				{
					printf("tid: %d particle terminated\n", tid);
				}*/
				
				local_counter++;	// increment # photons terminated

				// re-initialize all the arrays
				dcos.z = (ranecu(&seed) * 2.0f) - 1.0f;	

				rr = sqrt(1.0f - dcos.z*dcos.z);
				theta = ranecu(&seed) * twopipen;	
	
				dcos.x = rr*cos(theta);
				dcos.y = rr*sin(theta);

				if (((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) < (1.0f - epsilon)) || ((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) > (1.0f + epsilon)))
				{
					dcos.x = dcos.x/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
					dcos.y = dcos.y/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
					dcos.z = dcos.z/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
				}

				pos.x = info[tid].str_x; pos.y = info[tid].str_y; pos.z = info[tid].str_z; // set starting location of photon
				normal.x = 0.0f; normal.y = 0.0f; normal.z = 0.0f;
	
				if(beta > 0.0f)
					RoughSurface(&normal, &seed, beta);	// perturb smooth normal according to 'beta'

				local_num_generated++;
				absorbed = 0;
				detect = 0;
				bulk_abs = 0;
				num_rebound = 0;
				Xc = 0.0f;
				Yc = 0.0f;

			}
			else if( ((detect == 1) || (absorbed == 1) || (bulk_abs == 1)) && (local_counter == (NUM_EACH_THREAD-1)) )  // all the photons transported for this thread
			{
				/*if(tid == 6 || tid == 7)
				{
					printf("tid: %d all photons transported\n", tid);
				}*/
				local_counter++;
				break;
			}
			else if( (detect == 0) && (absorbed == 0) && (bulk_abs == 0) && (fabs(dcos.z - 0.0f) < epsilon) )  // check for trapped particle going back & forth dcos(z)=0
			{
				/*if(tid == 6 || tid == 7)
				{
					printf("tid: %d ccehck for trapped particles\n", tid);
				}*/
				
				// kill the particle and generate a new one instead - do not increment the local_counter
				// re-initialize all the arrays
		 		dcos.z = (ranecu(&seed) * 2.0f) - 1.0f;	

				rr = sqrt(1.0f - dcos.z*dcos.z);
				theta = ranecu(&seed) * twopipen;	
	
				dcos.x = rr*cos(theta);
				dcos.y = rr*sin(theta);

				if (((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) < (1.0f - epsilon)) || ((sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z)) > (1.0f + epsilon)))
				{
					dcos.x = dcos.x/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
					dcos.y = dcos.y/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
					dcos.z = dcos.z/(sqrt(dcos.x*dcos.x + dcos.y*dcos.y + dcos.z*dcos.z));
				}

				pos.x = info[tid].str_x; pos.y = info[tid].str_y; pos.z = info[tid].str_z;
				normal.x = 0.0f; normal.y = 0.0f; normal.z = 0.0f;
	
				if(beta > 0.0f)
					RoughSurface(&normal, &seed, beta);

				absorbed = 0;
				detect = 0;
				bulk_abs = 0;
				num_rebound = 0;
				Xc = 0.0f;
				Yc = 0.0f;
			}
			else	// transport photon
			{
				/*if(tid == 6 || tid == 7)
				{
					printf("tid: %d transport photon\n", tid);
				}*/
				num_rebound++;	// increment number of times the photon is rebounded from surface walls of a column
			    absorbed = boundary_analysis(&normal, &pos, &dcos, &seed, xdetector, ydetector, R, H, n1, n2, top_absfrac, beta, 
			    d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, &Xc, &Yc, pitch, info, tid, num_detected_primary, 
			    d_max, sensorRefl, dPhotonHist, boolToCollectData);
			}
		} 	// while loop ends

		atomicAdd(&num_generated, local_num_generated);

	}	// if tid=rowsread condition ends
	return;
}
#else		// C code

	int algo(float *normal, float *old_pos, float *pos, float *dcos, unsigned long long int *num_rebound, int* seed, struct start_info info, unsigned long long int *myimage, 
	float xdetector, float ydetector, float R, float H, float n1, float n2, float top_absfrac, float bulk_abscoeff, float beta, float d_min, int pixelsize, float lbound_x, 
	float lbound_y, float ubound_x, float ubound_y, float sensorRefl, float d_max, int ydim, int *h_num_detected_prim)
	{

		float rr=0.0f, theta=0.0f, norm = 0.0f;		// used in calculating directional cosines
		float rnd_num = 0.0f;
		int myresult = 0;				// flag to check if the photon is terminated (yes=1; no=0)

		if(absorbed == 0)   		// not absorbed at the top surface of detector, check for absorption in the bulk and detection       
		 {
			bulk_abs = isotropic(pos, dcos, seed, bulk_abscoeff, R, H, xdetector, ydetector, info, num_rebound[local_counter]);

			if(bulk_abs == 0)	// not absorbed in the bulk, check for detection
			{
				detect = detection(pos, H, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, sensorRefl, d_min, seed, dcos, 
				normal, bulk_abscoeff, R, xdetector, ydetector, num_rebound[local_counter], ydim, h_num_detected_prim);
			}
		 }

		 
		if( (detect == 1) || (absorbed == 1) || (bulk_abs == 1) )	// photon terminated
		 {
			local_counter++;				// increment the photon counter

			// calculate directional cosines
			rnd_num = (ranecu(seed) * 2.0f) - 1.0f; 	// generate random number between (-1,1)	 	

			dcos[2] = rnd_num;				// generate random number between (-1,1)
			rr = sqrt(1.0-rnd_num*rnd_num);
			theta=ranecu(seed)*twopipen;			// generate random number between 0 and 2pi
			dcos[0]=rr*cos(theta);
			dcos[1]=rr*sin(theta);

			// normalize
			norm = sqrt(dcos[0]*dcos[0] + dcos[1]*dcos[1] + dcos[2]*dcos[2]);

			if ((norm < (1.0f - epsilon)) || (norm > (1.0f + epsilon)))
			 {
				dcos[0] = dcos[0]/norm;
				dcos[1] = dcos[1]/norm;
				dcos[2] = dcos[2]/norm;
			 }


			pos[0] = info.str_x; pos[1] = info.str_y; pos[2] = info.str_z;	// set starting location of photon based on the PENELOPE scintillation events buffer
			old_pos[0] = info.str_x; old_pos[1] = info.str_y; old_pos[2] = info.str_z;
			normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 0.0f;		// initialize normal vector
	
			if(beta > 0.0f)
				RoughSurface(normal, seed, beta);	// perturb smooth normal according to 'beta' 

			absorbed = 0;
			detect = 0;
			bulk_abs = 0;

			myresult = 1;		// flag for photon termination

		 }
		else if( (detect == 0) && (absorbed == 0) && (bulk_abs == 0) && (fabs(dcos[2] - 0.0f) < epsilon) )  // check for trapped photon going back and forth dcos(z)=0
		 {
			// kill the photon and generate a new one instead - do not increment the counter

			// re-compute directional cosines
			rnd_num = (ranecu(seed) * 2.0f) - 1.0f; 
		 	
			dcos[2] = rnd_num;		
			rr = sqrt(1.0-rnd_num*rnd_num);
			theta=ranecu(seed)*twopipen;
			dcos[0]=rr*cos(theta);
			dcos[1]=rr*sin(theta);

			norm = sqrt(dcos[0]*dcos[0] + dcos[1]*dcos[1] + dcos[2]*dcos[2]);

			if ((norm < (1.0f - epsilon)) || (norm > (1.0f + epsilon)))
			 {
				dcos[0] = dcos[0]/norm;
				dcos[1] = dcos[1]/norm;
				dcos[2] = dcos[2]/norm;
			 }

			pos[0] = info.str_x; pos[1] = info.str_y; pos[2] = info.str_z;
			normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 0.0f;
	
			if(beta > 0.0f)
				RoughSurface(normal, seed, beta);	

			absorbed = 0;
			detect = 0;
			bulk_abs = 0;

			myresult = 0;	// photon still alive
		 }
		else
		 {
			num_rebound[local_counter]++;	// increment the number of rebounds this photon undergoes from columnar walls
		    	absorbed = boundary_analysis(normal, pos, dcos, seed, xdetector, ydetector, R, H, n1, n2, top_absfrac, beta, d_min, 
		    	pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, d_max, sensorRefl, ydim, h_num_detected_prim);

			myresult = 0;	// photon still alive
		 }

	return myresult;

	}

#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// determine where photon hits next within the column or if it gets absorbed in the material
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA

	__device__ inline int isotropic(float3 *pos, float3 *dcos, int2* seed, float bulk_abscoeff, float R, float H, float xdetector, float ydetector,
 	struct start_info *info, unsigned long long int mynum_rebound, float *Xc, float *Yc, int mytid, struct histStruct * dPhotonHist, int boolToCollectData)
	{
		float dsurf = 999.0f;		// distance to surface walls
		float dabs = 999.0f;		// distance to bulk absorption
		int flag_bulkabs = 0;		// flag to check if photon absorbed in the bulk
		
		dsurf = dist_to_surface(pos, dcos, R, H, xdetector, ydetector, info, mynum_rebound, Xc, Yc, mytid, dPhotonHist, boolToCollectData);	// distance to surface
		
		if (bulk_abscoeff > 0.0f)	
			dabs = (-1.0f/bulk_abscoeff) * log(ranecu(seed));					// distance to bulk absorption
		else
			dabs = 999999.0f;

		if (fabs(dsurf-(-99.0f)) < epsilon)				// photon lost: goes out of detector boundaries in dist_to_surface() function
		{
			flag_bulkabs = 1;
		}
		else if ( (dsurf < dabs) && (dsurf >= 0.0f) )			// photon not absorbed
		 {		
			flag_bulkabs = 0;
		 }
		else if ( (dsurf >= dabs) && (dabs >= 0.0f) )			// photon absorbed
		 {
			flag_bulkabs = 1;

			atomicAdd(&num_abs_bulk,1);				// increment to the number of optical photons absorbed in the bulk
			
			if(mytid < dev_numPhotonHist && boolToCollectData == 1)
			{
				dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
				dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
				dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
				dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
				dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
				dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 4;
				dPhotonHist[mytid].histCounter ++;
			}
		 }

	   return flag_bulkabs;
	}

#else	// C code

	int isotropic(float *pos, float *dcos, int* seed, float bulk_abscoeff, float R, float H, float xdetector, float ydetector, 
	struct start_info info, unsigned long long int mynum_rebound)
	{
		float dsurf = 999.0f;		// distance to surface
		float dabs = 999.0f;		// distance to bulk absorption
		int flag_bulkabs = 0;		// flag to check if photon absorbed in the bulk

		dsurf = dist_to_surface(pos, dcos, R, H, xdetector, ydetector, info, mynum_rebound);	// distance to surface

		if (bulk_abscoeff > 0.0f)	
			dabs = (-1.0f/bulk_abscoeff) * log(ranecu(seed));				// distance to absorption
		else
			dabs = 999999.0f;

		if (fabs(dsurf-(-99.0f)) < epsilon)				// photon lost: goes out of detector boundaries in dist_to_surface() function
		{
			flag_bulkabs = 1;
		}
		else if ( (dsurf < dabs) && (dsurf >= 0.0f) )			// photon not absorbed
		 {		
			flag_bulkabs = 0;
		 }
		else if ( (dsurf >= dabs) && (dsurf >= 0.0f) )			// photon absorbed
		 {
			flag_bulkabs = 1;

			num_abs_bulk++;						// increment to the number of optical photons absorbed in the bulk
		 }

	   return flag_bulkabs;
	}
#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// calculate distance to surface (within same column)
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA

	__device__ float dist_to_surface(float3 *pos, float3 *dcos, float R, float H, float xdetector, float ydetector, struct start_info *info, 
	unsigned long long int mynum_rebound, float *Xc, float *Yc, int mytid, struct histStruct * dPhotonHist, int boolToCollectData)
	{
		float d=999.0f;				// distance to surface walls
		float d1=999.0f, d2=999.0f;
		float d_plane=999.0f, d_cyl=999.0f;	// distance to z-plane; to surface walls of the cylinder column
		float3 temp_pos = {0.0f};
		float3 my1 = {0.0f};
		float R1 = 999.0f, R2 = 999.0f;
		float stepsize = 0.5f;			// used for moving the photon towards the column, if it goes outside due to precision errors.
		int repeat = 0, ctr1 = 0; 		// number of times photon should be moved in steps towards the column before killing. In this code maximum steps = 100.
							// Valid only when goes out of column.
		
		/*if(mytid == 6 || mytid == 7)
		{
			printf("tid: %d in dist_to_surface\n", mytid);
		}*/
		
		// store current position in a temporary variable
		temp_pos.x = pos->x;
		temp_pos.y = pos->y;
		temp_pos.z = pos->z;

		// center of first column (assumed as x,y coordinates of the energy deposition event from Penelope)
		if(mynum_rebound == 0)				
		{
			*Xc = info->str_x;
			*Yc = info->str_y;
		}

		// solving quadratic equation for distance from a point to the surface of cylinder
		my1.x = dcos->x*dcos->x + dcos->y*dcos->y;
		my1.y = 2.0f*( ((pos->x-(*Xc))*dcos->x) + ((pos->y-(*Yc))*dcos->y) );
		my1.z = (pos->x-(*Xc))*(pos->x-(*Xc)) + (pos->y-(*Yc))*(pos->y-(*Yc)) - (R*R);
	
		// actual distance d = (d1 or d2)/sin_theta2	
		d1 = (-my1.y + (sqrt( (my1.y*my1.y) - (4.0f*my1.x*my1.z) )))/(2.0f * my1.x);
		d2 = (-my1.y - (sqrt( (my1.y*my1.y) - (4.0f*my1.x*my1.z) )))/(2.0f * my1.x);


		// hits either upper half surface or top of the cylinder
		if(dcos->z > 0.0f) 
		 {

		  if((fabs(dcos->x - 0.0f) < epsilon) && (fabs(dcos->y - 0.0f) < epsilon) && (fabs(dcos->z - 1.0f) < epsilon))  
		  // if photon travel straight in +z axis direction
		   {
			d = (H/2.0f - pos->z)/dcos->z;

			pos->z = H/2.0f;	

			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;
		   }
		  else
		   {		

			// calculate distance to infinite plane at z=H/2
			d_plane = (H/2.0f - pos->z)/(dcos->z);

			// calculate the distance to the upper half of the cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}

			// find min from d_plane and d_cyl
			if(d_plane >= d_cyl)
			 {
				d = d_cyl;
				pos->z = temp_pos.z + d*dcos->z;
			 }
			else
			 {
				d = d_plane;		
				pos->z = H/2.0f;
			 }

			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;

		   }	// else loop ends
	
		
		 } // if loop for dcos.z > 0 ends

		else if(dcos->z < 0.0f) // hits either lower half or bottom of cylinder
		 {
		  // if photon travels in -Z direction staright, then it should get detected
		  if ((fabs(dcos->x-0.0f) < epsilon) && (fabs(dcos->y-0.0f) < epsilon) && (fabs(dcos->z - (-1.0f)) < epsilon))  
		   {
			d = (-H/2.0f - pos->z)/dcos->z;  
			pos->z = -H/2.0f;
				
			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;
		   }
		  else
		   {

			// calculate distance to infinite plane at z=-H/2
			d_plane = (-H/2.0f - pos->z)/(dcos->z);

			// calculate the distance to the lower half of the cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}
		

			// find min from d_plane and d_cyl
			if(d_plane >= d_cyl)
			 {
				d = d_cyl;
				pos->z = temp_pos.z + d*dcos->z;
			 }
			else
			 {
				d = d_plane;		
				pos->z = -H/2.0f;
			 }

			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;

		   }	// else loop ends

		 }	// else if loop for dcos.z < 0 ends

		else	// when dcos.z=0.0 (will hit only the side of the cylinder)
		 {
			// calculate the distance to the side of cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}
		
		
			d = d_cyl;
			pos->z = temp_pos.z + d*dcos->z;

			pos->x = temp_pos.x + d*dcos->x;
			pos->y = temp_pos.y + d*dcos->y;

		 }

		// condition to check that pos is within detector boundaries - if true, photon LOST
		if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) || (pos->z < -H/2.0f) || (pos->z > H/2.0f)  )
			{
				d = -99.0f;
				atomicAdd(&num_lost,1);
				goto distexit;
			}
		else
			atomicAdd(&photon_distance, d);		// add distance travelled to global variable


		// check if photon is outside the current column. This can happen due to single precision errors. If yes, then move the photon towards the column based on the 'stepsize' and check again. Repeat this 100 times, if still outside then kill the photon.
		R1 = sqrt((pos->x - (*Xc))*(pos->x - (*Xc)) + (pos->y - (*Yc))*(pos->y - (*Yc)));

		repeat = 0;	// counters for checking how many times has the photon been moved towards the column.
		ctr1 = 0;

		while( (R1 > (R-1e-5)) && (repeat < 10) && (ctr1 < 10) ) // R1 > R-some small value; to avoid single precision errors
		{

			// store current position
			temp_pos.x = pos->x;
			temp_pos.y = pos->y;

			// move photon by stepsize (in microns) in the incident direction
			pos->x = pos->x + stepsize*(-dcos->x);
			pos->y = pos->y + stepsize*(-dcos->y);

			R2 = sqrt((pos->x - (*Xc))*(pos->x - (*Xc)) + (pos->y - (*Yc))*(pos->y - (*Yc))); // new radius

			if(R2 > R1) // means the photon is moving farther away from the column 
			{	    // this can happen if the stepsize is big enough for the photon to pass through the column and exit on the other side.

				// move it back to previous position, reduce the stepsize and try moving it again.
				pos->x = temp_pos.x;
				pos->y = temp_pos.y;

				stepsize = stepsize/2.0f;
				ctr1++;
			}
			else
			{
				R1 = R2;
				repeat++;
			}

		}

		// kill the photon if still outside the column
		if(R1 > (R-1e-5))
		 {
			d = -99.0f;
			atomicAdd(&num_outofcol,1);
			goto distexit;
		 }

		// condition to check that pos is within detector boundaries - if true, photon LOST
		if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) || (pos->z < -H/2.0f) || (pos->z > H/2.0f)  )
			{
				d = -99.0f;
				atomicAdd(&num_lost,1);
				goto distexit;
			}

	distexit:
		if(mytid < dev_numPhotonHist && boolToCollectData == 1)
		{
			if(d == -99.0f)
			{
				//photon lost
			}
			else
			{
				dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
				dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
				dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
				dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
				dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
				dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
				dPhotonHist[mytid].histCounter ++;
				
				/*if(mytid == 6 || mytid == 7)
				{
					printf("dist_to_surface - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
				}*/
			}
		}
		return d;
	}	

#else	// C code

	float dist_to_surface(float *pos, float *dcos, float R, float H, float xdetector, float ydetector, struct start_info info, 
	unsigned long long int mynum_rebound)
	{
		float d=999.0f;				// distance to surface walls
		float d1=999.0f, d2=999.0f;
		float d_plane=999.0f, d_cyl=999.0f;	// distance to z-plane; to surface walls of the cylinder column
		float temp_pos[3] = {0.0f};
		float my1[3] = {0.0f};
		float R1 = 999.0f, R2 = 999.0f;
		float stepsize = 0.5f;			// used for moving the photon towards the column, if it goes outside due to precision errors.
		int repeat = 0, ctr1 = 0; 		// number of times photon should be moved in steps towards the column before killing. In this code maximum steps = 100.
							// Valid only when goes out of column.

		// store current position in a temporary variable
		temp_pos[0] = pos[0];
		temp_pos[1] = pos[1];
		temp_pos[2] = pos[2];

		// center of first column (assumed as x,y coordinates of the energy deposition event from Penelope)
		if(mynum_rebound == 0)				
		{
			Xc = info.str_x;
			Yc = info.str_y;
		}

		// solving quadratic equation for distance from a point to the surface of cylinder
		my1[0] = dcos[0]*dcos[0] + dcos[1]*dcos[1];
		my1[1] = 2.0f*( ((pos[0]-(Xc))*dcos[0]) + ((pos[1]-(Yc))*dcos[1]) );
		my1[2] = (pos[0]-(Xc))*(pos[0]-(Xc)) + (pos[1]-(Yc))*(pos[1]-(Yc)) - (R*R);
	
		// actual distance d = (d1 or d2)/sin_theta2	
		d1 = (-my1[1] + (sqrt( (my1[1]*my1[1]) - (4.0f*my1[0]*my1[2]) )))/(2.0f * my1[0]);
		d2 = (-my1[1] - (sqrt( (my1[1]*my1[1]) - (4.0f*my1[0]*my1[2]) )))/(2.0f * my1[0]);

	
		// hits either upper half surface or top surface of cylinder
		if(dcos[2] > 0.0f) 
		 {

		  if((fabs(dcos[0] - 0.0f) < epsilon) && (fabs(dcos[1] - 0.0f) < epsilon) && (fabs(dcos[2] - 1.0f) < epsilon))  
		  // if photon travel straight in +z axis direction
		   {
			d = (H/2.0f - pos[2])/dcos[2];

			pos[2] = H/2.0f;	

			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];
		   }
		  else
		   {		

			// calculate distance to infinite plane at z=H/2
			d_plane = (H/2.0f - pos[2])/(dcos[2]);

			// calculate the distance to the upper half of the cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}

			// find min from d_plane and d_cyl
			if(d_plane >= d_cyl)
			 {
				d = d_cyl;
				pos[2] = temp_pos[2] + d*dcos[2];
			 }
			else
			 {
				d = d_plane;		
				pos[2] = H/2.0f;
			 }

			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];

		   }	// else loop ends
	
		
		 } // if loop for dcos[2] > 0 ends

		else if(dcos[2] < 0.0f) // hits either lower half or bottom of cylinder
		 {
		  // if photon travels in -Z direction staright, then it should get detected
		  if ((fabs(dcos[0]-0.0f) < epsilon) && (fabs(dcos[1]-0.0f) < epsilon) && (fabs(dcos[2] - (-1.0f)) < epsilon))  
		   {
			d = (-H/2.0f - pos[2])/dcos[2];  
			pos[2] = -H/2.0f;
				
			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];
		   }
		  else
		   {

			// calculate distance to infinite plane at z=-H/2
			d_plane = (-H/2.0f - pos[2])/(dcos[2]);

			// calculate the distance to the lower half of the cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}
		

			// find min from d_plane and d_cyl
			if(d_plane >= d_cyl)
			 {
				d = d_cyl;
				pos[2] = temp_pos[2] + d*dcos[2];
			 }
			else
			 {
				d = d_plane;		
				pos[2] = -H/2.0f;
			 }

			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];

		   }	// else loop ends

		 }	// else if loop for dcos[2] < 0 ends

		else	// when dcos[2]=0.0 (will hit only the side of the cylinder)
		 {
			// calculate the distance to the side of cylinder
			if(d1 >= d2) 
			{
				d_cyl = d1;
			}
			else if(d2 > d1)
			{
				d_cyl = d2;
			}
		
		
			d = d_cyl;
			pos[2] = temp_pos[2] + d*dcos[2];

			pos[0] = temp_pos[0] + d*dcos[0];
			pos[1] = temp_pos[1] + d*dcos[1];

		 }

		// condition to check that pos is within detector boundaries - if true, photon LOST
		if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) || (pos[2] < -H/2.0f) || (pos[2] > H/2.0f)  )
			{
				d = -99.0f;
				num_lost++;
				goto distexit;
			}
		else
			photon_distance = photon_distance + d;		// add distance travelled to global variable


		// check if photon is outside of current column. This can happen due to single precision errors. If yes, then move the photon towards the column based on the 'stepsize' and check again. Repeat this 100 times, if still outside then kill the photon.
		R1 = sqrt((pos[0] - Xc)*(pos[0] - Xc) + (pos[1] - Yc)*(pos[1] - Yc));
		
		repeat = 0;
		ctr1 = 0;

		while( (R1 > (R-1e-5)) && (repeat < 10) && (ctr1 < 10) ) // R1 > R1-some small value..because of single precision errors that comparison with R may generate
		{

			// store current position
			temp_pos[0] = pos[0];
			temp_pos[1] = pos[1];

			// move photon by 0.5 um in the incident direction
			pos[0] = pos[0] + stepsize*(-dcos[0]);
			pos[1] = pos[1] + stepsize*(-dcos[1]);

			R2 = sqrt((pos[0] - Xc)*(pos[0] - Xc) + (pos[1] - Yc)*(pos[1] - Yc));

			if(R2 > R1) // means the photon is moving farther away from the column 
			{	    // this can happen if the stepsize is big enough for the photon to pass through the column and exit on other side.

				// move it back to previous position, reduce the stepsize and try moving it again.
				pos[0] = temp_pos[0];
				pos[1] = temp_pos[1];

				stepsize = stepsize/2.0f;
				ctr1++;
			}
			else
			{
				R1 = R2;
				repeat++;
			}

		}

		// kill the photon if still outside the column
		if(R1 > (R-1e-5))
		 {
			d = -99.0f;
			num_outofcol++;
			goto distexit;
		 }

		// condition to check that pos is within detector boundaries - if true, photon LOST
		if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) || (pos[2] < -H/2.0f) || (pos[2] > H/2.0f)  )
			{
				d = -99.0f;
				num_lost++;
				goto distexit;
			}

	distexit:
	 return d;
	}
	
#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// calculate the directional cosines of the reflected vector
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
__device__ int boundary_analysis(float3 *normal, float3 *pos, float3 *dcos, int2* seed, float xdetector, float ydetector, float R, float H, float n1, float n2, 
float top_absfrac, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, 
float *Xc, float *Yc, size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float d_max, float sensorRefl, struct histStruct * dPhotonHist,
int boolToCollectData)
{
	float3 dcos_temp = {0.0f};
	float Pr = 0.0f, Pt = 0.0f;	// Probability of reflection and transmission
	float theta1 = 0.0f;		// angle between normal and reflected (radians)
	float theta2 = 0.0f;		// angle between normal and transmitted (radians)
	float cct1 = 0.0f;		// columnar crosstalk
	int trans_flag = 0.0f;		// flag - photon terminated during transmission to a new column; used in transmit() (terminated: yes=1; no=0)
	int flag_abs = 0;		// flag - photon absorbed at the top surface (yes=1; no=0)
	int flag_call_transmit = 1;	// flag - photon moves within a column (flag = 0) [call isotropic()] or between columns (flag = 1) [call transmit()]
	int flagCCT = 0;		// flag - photon cross over (yes=1; no=0); send as input to transmit()
	int newnormalctr=0;		// counter - if angle between inverted dir. cosine and rough normal > 1.57 or < 0 radians (recalculate normal; max. 100 times)
	int theta1ctr=0;		// counter - if theta1 > 1.57 or < 0 radians (recalculate normal; max. 100 times)
	int oldN_Rctr=0;		// counter - if angle between reflected dir. cosine and smooth normal > 1.57 radians (recalculate max. 25 times)
	int reperturb_ctr = 0;		// counter - if angle between reflected dir. cosine and rough normal > 1.57 radians (reperturb normal; max. 3 times)
	float newdepth = 0.0f;		// bottom depth for which CCT=1 (z_a)
	float temp_norm = 0.0f;
	float mag = 0.0f;
	float rr_rnd = 0.0f, theta_rnd = 0.0f;
	float3 old_normal = {0.0f};
	float3 old_dcos = {0.0f};
	float angle_oldN_R = 0.0f;
	float cos_newangle = 0.0f, newangle = 0.0f;

	/*if(mytid == 6 || mytid == 7)
	{
			printf("tid: %d in boundary_analysis\n", mytid);
	}*/
	
	// determine the coordinates of normal
	if ( (fabs(pos->z - (float)(H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	// reached top surface and dir. cosine in z-direction is positive
	{
		if ( (top_absfrac > 0.0f) && (ranecu(seed) < top_absfrac) )	// photon gets absorbed; 'top_absfrac' is the top surface absorption fraction (0,1)
		{
			flag_abs = 1;
			atomicAdd(&num_abs_top, 1);				// increment # photons absorbed at the top

			if(mytid < dev_numPhotonHist && boolToCollectData == 1)
			{
				dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
				dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
				dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
				dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
				dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
				dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 3;
				dPhotonHist[mytid].histCounter ++;
			}
			
			/*if(mytid == 6 || mytid == 7)
			{
				printf("boundary analysis - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
			}*/
		}
		else
		{
			// normal		
			normal->x = 0.0f;
			normal->y = 0.0f;
			normal->z = -1.0f;

			// assign new directional cosines - top surface being isotropic reflector
			dcos->z = -fabs((ranecu(seed) * 2.0f) - 1.0f);
			rr_rnd = sqrt(1.0f - dcos->z*dcos->z);
			theta_rnd = ranecu(seed)*twopipen;	

			dcos->x=rr_rnd*cos(theta_rnd);
			dcos->y=rr_rnd*sin(theta_rnd);
	
			flag_abs = 0;
		}
	}	
	else 	// gets reflected or transmitted
	{	
		// Columnar crosstalk
		newdepth = H*0.2f;	// top 20% depth CCT=1. considering CsI layer only. NO organic polymer coating.
			
		if( (pos->z <= H/2.0f) && (pos->z >= (H/2.0f - newdepth)) )	// top 20% - 100% cct
		{
			cct1 = 1.0f;
		}
		else if( (pos->z < (H/2.0f - newdepth)) && (pos->z >= 0.0f) )  // from 20% depth to 50% - linear 100% to 50% 
		{
			cct1 = (pos->z/(2.0f*(H/2.0f - newdepth))) + 0.5;	
		}
		else if( (pos->z < 0.0f) && (pos->z >= (-H/2.0f)) ) // bottom 50% to -H/2 - 50% to 100% CCT
		{
			cct1 = ( (pos->z - (-H/2.0f))/(2.0f * (-H/2.0f)) ) + 1.0 ;
		}
	
		if(ranecu(seed) < cct1)		// columnar crosstalk occurs
		{
			// photon crosses over to adjacent column with random orientation. directional cosine do not change.
			flagCCT = 1;

			trans_flag = transmit(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, 
			lbound_x, lbound_y, ubound_x, ubound_y, myimage, pitch, info, mytid, num_detected_primary, d_max, sensorRefl, 
			flagCCT, Xc, Yc, dPhotonHist, boolToCollectData);

			if (trans_flag == 1)		// photon terminated during transmission
				flag_abs = 1;
			else if (trans_flag == 0)	// photon still alive; transmitted to adjacent column
			{
				// calculate new column's center coordinates
				*Xc = (float)( pos->x + R*(-normal->x) );
				*Yc = (float)( pos->y + R*(-normal->y) );

				if(mytid < dev_numPhotonHist && boolToCollectData == 1)
				{
					dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
					dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
					dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
					dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
					dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
					dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
					dPhotonHist[mytid].histCounter ++;
				}
				
				/*if(mytid == 6 || mytid == 7)
				{
					printf("boundary analysis - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
				}*/
				flag_abs = 0;
			}
		}
		else	// no cross over
		{
			prpt:
				// within the column
				if(flag_call_transmit == 1)			// photon is currrently within a column with center Xc,Yc
				{
					mag = sqrt( (((*Xc)-pos->x) * ((*Xc)-pos->x)) + (((*Yc)-pos->y) * ((*Yc)-pos->y)) );
					normal->x = ((*Xc)-pos->x)/mag;
					normal->y = ((*Yc)-pos->y)/mag;
					normal->z = 0.0f;
		
					if(beta > 0.0f)
						RoughSurface(normal, seed, beta);	// perturb normal for rough surface

					flag_abs = 0;
				}
				// outside the column
				else if (flag_call_transmit == 0)		// photon is currently between columns and has not entered any column yet. New normal is sampled in the transmit(), so do not calculate normal here.
				{
					// center of new column (obtained by inverting the new normal sampled in transmit() and finding center at distance R from current position)
					*Xc = (float)( pos->x + R*(-normal->x) );
					*Yc = (float)( pos->y + R*(-normal->y) );

					if(mytid < dev_numPhotonHist && boolToCollectData == 1)
					{
						dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
						dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
						dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
						dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
						dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
						dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
						dPhotonHist[mytid].histCounter ++;
					}
					
					/*if(mytid == 6 || mytid == 7)
					{
						printf("boundary analysis - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
					}*/
			
				    flag_abs = 0;
				}

				
				dcos_temp.x = -dcos->x;		// -dcos -> invert the incident dcos vector; to get the smaller angle between normal and dcos
				dcos_temp.y = -dcos->y;
				dcos_temp.z = -dcos->z;

				old_normal.x = normal->x;
				old_normal.y = normal->y;
				old_normal.z = normal->z;
	
				old_dcos.x = dcos->x;
				old_dcos.y = dcos->y;
				old_dcos.z = dcos->z;

			reperturb:
				normal->x = old_normal.x;
				normal->y = old_normal.y;
				normal->z = old_normal.z;

				dcos->x = old_dcos.x;
				dcos->y = old_dcos.y;
				dcos->z = old_dcos.z;

				dcos_temp.x = -dcos->x;
				dcos_temp.y = -dcos->y;
				dcos_temp.z = -dcos->z;

				if( (flag_call_transmit == 1) && (reperturb_ctr != 0) )		// within the column
				{
					if(beta > 0.0f)
						RoughSurface(normal, seed, beta);	
				}
				if( (flag_call_transmit == 0) && (reperturb_ctr != 0) )		// outside the column
				{
					if(beta > 0.0f)
						RoughSurface(normal, seed, beta);	

					// center of new column (obtained by inverting the new normal sampled in transmit() and finding center at distance R from current position)
					*Xc = (float)( pos->x + R*(-normal->x) );
					*Yc = (float)( pos->y + R*(-normal->y) );

					if(mytid < dev_numPhotonHist && boolToCollectData == 1)
					{
						dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
						dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
						dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
						dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
						dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
						dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
						dPhotonHist[mytid].histCounter ++;
					}
					
					/*if(mytid == 6 || mytid == 7)
					{
						printf("boundary analysis - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
					}*/
				}

				// Using Snell's law, calculate theta1 (angle between normal and reflected) and theta2 (angle between normal and transmitted)
			no_perturbation:
				theta1 = dot_product(&dcos_temp,normal);		// cosine of angle between incident in opposite direction and normal (in radians)

				if ( (theta1 > 1.0f) || (theta1 < 0.0f) )	// if incidence angle > 1.57 radian or < 0 radian, then recalculate normal
				{
					// if photon was transmitted, then new normal has to be sampled again
					if(flag_call_transmit == 0)
					{
					mynewnormal:
						normal->x = dcos_temp.x;		// normal = inverted dcos of incident vector
						normal->y = dcos_temp.y;
						normal->z = dcos_temp.z;

						RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

						mag = sqrt(normal->x*normal->x + normal->y*normal->y);

						// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
						normal->z = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
						normal->x = normal->x/mag;		
						normal->y = normal->y/mag;

						if(beta > 0.0f)
							RoughSurface(normal, seed, beta);

						// find the angle between normal and -dcos
						cos_newangle = dot_product(&dcos_temp, normal);
						newangle = acosf(cos_newangle);

						if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new rough normal is within +- 1.57 radians from inverted dcos
						{						// keep looping until get a theta within 1.57 radians - maximum iterations 100
							if(newnormalctr < 100)
							{
								newnormalctr++;
								goto mynewnormal;			
							}
							else 					// else terminate the photon
							{
								atomicAdd(&num_theta1,1);  // increment the counter - # photons terminated due to incidence angle > 1.57 or < 0 radian
								flag_abs = 1;
								newnormalctr = 0;
								goto baexit;
							}
						}

					}
		
					if(theta1ctr < 100)	// recalculate max. 100 times	
					{
						theta1ctr++;
						goto prpt;
					}
					else			// terminate photon
					{
						atomicAdd(&num_theta1,1);	
						flag_abs = 1;
						theta1ctr = 0;
						goto baexit;
					}
				}
				else				// 0 < theta1 < 1.57 (radian); continue the photon transport
					theta1 = acosf(theta1);
		

				// check for conditions where photon can only reflect
				if (flag_call_transmit == 1)	// only valid when photon is within the column and can transmit outside the column. asin(n1/n2) -> nan
				{
					if (theta1 > asin(n2/n1))	// critical angle condition for total internal reflection (TIR)
					{
						Pr = 1.0f;		// TIR occurs
						Pt = 0.0f;
					}
			       		else if ( theta1 < epsilon ) 	// theta1 ~= 0, always reflect
					{
				        	theta1 = 0.00042;       			// make theta1 = very small number, to avoid getting nan probabilities
					        theta2 = asinf((float)(n1/n2)*sin(theta1));     // refracted/transmitted angle in radians

					        // Using Fresnel's law, compute probability of reflection and transmission 
					        Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
				        	Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
					else    // the ray will transmit
					{
				        	theta2 = asinf((float)(n1/n2)*sin(theta1));     // refracted/transmitted angle in radians
					        
					        // Using Fresnel's law, compute probability of reflection and transmission 
				        	Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
					        Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}

				}
				else if (flag_call_transmit == 0)	// outside the column
				{		
					if((n1/n2) < 1.57f)	
					{
						if (theta1 > asin(n1/n2))	// critical angle condition for total internal reflection (TIR)
						{
							Pr = 1.0f;		// TIR occurs
							Pt = 0.0f;
						}
					}
					else if ( theta1 < epsilon )	// theta1 ~= 0, then always reflect
					{
						theta1 = 0.00042;	// make theta1 a very smal number, to avoid getting nan probabilities
						theta2 = asinf((float)(n1/n2)*sin(theta1)); 	// refracted/transmitted angle in radians
	
						// Using Fresnel's law, compute probability of reflection and transmission 
						Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
						Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
					else	// photon transmits
					{
						theta2 = asinf((float)(n2/n1)*sin(theta1));

						// Using Fresnel's law, compute probability of reflection and transmission 
						Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
						Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
				}


				// normalize Pr and Pt
				temp_norm = Pr + Pt;
				Pr = Pr/temp_norm;
				Pt = Pt/temp_norm;

				if(ranecu(seed) < Pr)				// reflection
				{
					trans_dir_cos(dcos, normal, theta1, theta2, 0, mytid, info);	// compute new directional cosines

					// check that reflected vector is within 1.57 radians from original normal
					angle_oldN_R = dot_product(&old_normal, dcos);
					angle_oldN_R = acosf(angle_oldN_R);


					if (angle_oldN_R > 1.57f) 		// > 1.57 radians, reperturb the normal
					{
						reperturb_ctr++;

						if(reperturb_ctr < 4)		// reperturb maximum 3 times
							goto reperturb;
						else				// else calculate using smooth surface normal (old_normal)
						{
							normal->x = old_normal.x;
							normal->y = old_normal.y;
							normal->z = old_normal.z;

							dcos->x = old_dcos.x;
							dcos->y = old_dcos.y;
							dcos->z = old_dcos.z;

							dcos_temp.x = -dcos->x;
							dcos_temp.y = -dcos->y;
							dcos_temp.z = -dcos->z;

							reperturb_ctr = 0;
				
							if(oldN_Rctr < 25)	// resample max 75 times (25 * reperturb 3 times)	
							{
								oldN_Rctr++;
								goto no_perturbation;
							}
							else			// terminate photon
							{
								atomicAdd(&num_theta1,1);// increment the counter - # photons terminated due to incidence angle > 1.57 or < 0 radian
								flag_abs = 1;
								oldN_Rctr = 0;
								goto baexit;
							}
						}
					}

					if (flag_call_transmit == 0)		// reflects between columns; calculate distance using transmit()
					{
						flagCCT = 0;	

						trans_flag = transmit(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, 
						lbound_x, lbound_y, ubound_x, ubound_y, myimage, pitch, info, mytid, num_detected_primary, d_max, sensorRefl, 
						flagCCT, Xc, Yc, dPhotonHist, boolToCollectData);

						if (trans_flag == 1)		// photon terminated
							flag_abs = 1;
						else if (trans_flag == 0)
							goto prpt;				
					}
				}
				else						// transmits 
				{
					trans_dir_cos(dcos, normal, theta1, theta2, 1, mytid, info);		// compute transmitted directional cosines

					if (flag_call_transmit == 1)		// photon travels between columns
					{
						flag_call_transmit = 0;
						flagCCT = 0;

						trans_flag = transmit(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, 
						lbound_x, lbound_y, ubound_x, ubound_y, myimage, pitch, info, mytid, num_detected_primary, d_max, sensorRefl, 
						flagCCT, Xc, Yc, dPhotonHist, boolToCollectData);

						if (trans_flag == 1)		// photon terminated
							flag_abs = 1;
						else if (trans_flag == 0)	// hits a column
							goto prpt;		// check again to see if it gets reflected or transmitted
					}			
				}
		
			} // else 'prpt ends
		
		} // main else ends

	baexit:
	   return flag_abs;

	}	

#else	// C code

	int boundary_analysis(float *normal, float *pos, float *dcos, int* seed, float xdetector, float ydetector, float R, float H, float n1, float n2, float top_absfrac, float beta, 	float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, float d_max, 
	float sensorRefl, int ydim, int *h_num_detected_prim)
	{

		float dcos_temp[3] = {0.0f};
		float Pr = 0.0f, Pt = 0.0f;	// Probability of reflection and transmission
		float theta1 = 0.0f;		// angle between normal and reflected vector (radians)
		float theta2 = 0.0f;		// angle between normal and transmitted vector (radians)
		float cct1 = 0.0f;		// columnar crosstalk
		int trans_flag = 0.0f;		// flag - photon terminated during transmission to a new column; used in transmit() (terminated: yes=1; no=0)
		int flag_abs = 0;		// flag - photon absorbed at the top surface (yes=1; no=0)
		int flag_call_transmit = 1;	// flag - photon moves within a column (flag = 0) [call isotropic()] or between columns (flag = 1) [call transmit()]
		int flagCCT = 0;		// flag - photon cross over (yes=1; no=0); send as input to transmit()
		int newnormalctr=0;		// counter - if angle between inverted dir. cosine and rough normal > 1.57 or < 0 radians (recalculate normal; max. 100 times)
		int theta1ctr=0;		// counter - if theta1 > 1.57 or < 0 radians (recalculate normal; max. 100 times)
		int oldN_Rctr=0;		// counter - if angle between reflected dir. cosine and smooth normal > 1.57 radians (recalculate max. 25 times)
		int reperturb_ctr = 0;		// counter - if angle between reflected dir. cosine and rough normal > 1.57 radians (reperturb normal; max. 3 times)
		float newdepth = 0.0f;		// bottom depth for which CCT=1 (z_a)
		float temp_norm = 0.0f;
		float mag = 0.0f;
		float rr_rnd = 0.0f, theta_rnd = 0.0f;
		float old_normal[3] = {0.0f};
		float old_dcos[3] = {0.0f};
		float angle_oldN_R = 0.0f;
		float cos_newangle = 0.0f, newangle = 0.0f;


		// determine the coordinates of normal
		if ( (fabs(pos[2] - (float)(H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	// reached top surface and dir. cosine in z-direction is positive
		{
	
			if ( (top_absfrac > 0.0f) && (ranecu(seed) < top_absfrac) )	// photon gets absorbed; 'top_absfrac' is the top surface absorption fraction (0,1)
			{
				flag_abs = 1;
				num_abs_top++;						// increment # photons absorbed at the top surface
			}
			else
			{
				normal[0] = 0.0f;
				normal[1] = 0.0f;
				normal[2] = -1.0f;

				// assign new directional cosines; top surface is isotropic reflector
				dcos[2] = -fabs((ranecu(seed) * 2.0f) - 1.0f);
				rr_rnd = sqrt(1.0f - dcos[2]*dcos[2]);
				theta_rnd = ranecu(seed)*twopipen;	

				dcos[0]=rr_rnd*cos(theta_rnd);
				dcos[1]=rr_rnd*sin(theta_rnd);
	
				flag_abs = 0;
			}

		}	
		else 	// photon reflected or transmitted
		{	
			// Columnar crosstalk
			newdepth = H*0.2f;	// top 20% depth CCT=1. considering CsI layer only. NO organic polymer coating.
		
			if( (pos[2] <= H/2.0f) && (pos[2] >= (H/2.0f - newdepth)) )	// top 20% - 100% cct
			{
				cct1 = 1.0f;
			}
			else if( (pos[2] < (H/2.0f - newdepth)) && (pos[2] >= 0.0f) )  // from 20% depth to 50% - linear 100% to 50% 
			{
				cct1 = (pos[2]/(2.0f*(H/2.0f - newdepth))) + 0.5;	
			}
			else if( (pos[2] < 0.0f) && (pos[2] >= (-H/2.0f)) ) // bottom 50% to (-H/2 - 4 um polymer) - 50% to 100% CCT
			{
				cct1 = ( (pos[2] - (-H/2.0f))/(2.0f * (-H/2.0f)) ) + 1.0 ;
			}

	
			if(ranecu(seed) < cct1)		// columnar crosstalk occurs
			{

				// photon crosses over to adjacent column with random orientation. directional cosine do not change.
				flagCCT = 1;

				trans_flag = transmit(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, d_max, sensorRefl, ydim, flagCCT, h_num_detected_prim);

				if (trans_flag == 1)		// photon terminated
					flag_abs = 1;
				else if (trans_flag == 0)	// photon still alive; crossed over to adjacent column
				{
					// calculate new column's center coordinates
					Xc = (float)( pos[0] + R*(-normal[0]) );
					Yc = (float)( pos[1] + R*(-normal[1]) );

					flag_abs = 0;
				}
			}
			else
			{

			prpt:

				// within the column
				if(flag_call_transmit == 1)			// photon is currently within a column with center Xc,Yc
				{
					mag = sqrt( (((Xc)-pos[0]) * ((Xc)-pos[0])) + (((Yc)-pos[1]) * ((Yc)-pos[1])) );
					normal[0] = ((Xc)-pos[0])/mag;
					normal[1] = ((Yc)-pos[1])/mag;
					normal[2] = 0.0f;
		
					if(beta > 0.0f)
						RoughSurface(normal, seed, beta);	// perturb normal for rough surface according to 'beta'

					flag_abs = 0;
				}
				// outside the column
				else if (flag_call_transmit == 0)		// photon is currently between columns and has not entered any column yet. New normal is sampled in the transmit(), so do not calculate normal here.
				{
					// center of new column (obtained by inverting the new normal sampled in transmit() and finding center at distance R from current position)
					Xc = (float)( pos[0] + R*(-normal[0]) );
					Yc = (float)( pos[1] + R*(-normal[1]) );

				        flag_abs = 0;
				}

				dcos_temp[0] = -dcos[0];	// -dcos -> invert the incident dcos vector; to get the smaller angle between normal and dcos
				dcos_temp[1] = -dcos[1];
				dcos_temp[2] = -dcos[2];

				old_normal[0] = normal[0];
				old_normal[1] = normal[1];
				old_normal[2] = normal[2];
	
				old_dcos[0] = dcos[0];
				old_dcos[1] = dcos[1];
				old_dcos[2] = dcos[2];

			reperturb:
				normal[0] = old_normal[0];
				normal[1] = old_normal[1];
				normal[2] = old_normal[2];

				dcos[0] = old_dcos[0];
				dcos[1] = old_dcos[1];
				dcos[2] = old_dcos[2];

				dcos_temp[0] = -dcos[0];
				dcos_temp[1] = -dcos[1];
				dcos_temp[2] = -dcos[2];

				if( (flag_call_transmit == 1) && (reperturb_ctr != 0) )		// within the column
				 {
					if(beta > 0.0f)
						RoughSurface(normal, seed, beta);	
				 }
				if( (flag_call_transmit == 0) && (reperturb_ctr != 0) )		// outside the column
				 {
					if(beta > 0.0f)
						RoughSurface(normal, seed, beta);	

					// center of new column (obtained by inverting the new normal sampled in transmit() and finding center at distance R from current position)
					Xc = (float)( pos[0] + R*(-normal[0]) );
					Yc = (float)( pos[1] + R*(-normal[1]) );

				 }
			
				// Using Snell's law, calculate theta1 (angle between normal and reflected) and theta2 (angle between normal and transmitted)
			no_perturbation:
				theta1 = dot_product(dcos_temp,normal);		// cosine of angle between incident in opposite direction and normal (in radians)

		
				if ( (theta1 > 1.0f) || (theta1 < 0.0f) )	// if incidence angle > 1.57 or < 0 radians, then recalculate normal
				{
					// if photon was transmitted, then new normal has to be sampled again
					if(flag_call_transmit == 0)
					{
					mynewnormal:
						normal[0] = dcos_temp[0];		// normal = inverted dir. cosine of the incident vector
						normal[1] = dcos_temp[1];
						normal[2] = dcos_temp[2];

						RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

						mag = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

						// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
						normal[2] = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
						normal[0] = normal[0]/mag;		
						normal[1] = normal[1]/mag;

						if(beta > 0.0f)
							RoughSurface(normal, seed, beta);

						// find the angle between normal and -dcos
						cos_newangle = dot_product(dcos_temp, normal);
						newangle = acosf(cos_newangle);

						if ( (newangle < 0.0f) || (newangle > 1.57f) )	// new normal within +- 1.57 radians from inverted dcos
						{						// keep looping until get a theta within 1.57 radians - maximum iterations 100
							if(newnormalctr < 100)
							{
								newnormalctr++;
								goto mynewnormal;			
							}
							else 					// else terminate photon
							{
								num_theta1++;
								flag_abs = 1;
								newnormalctr = 0;
								goto baexit;
							}				
						}

					}


					if(theta1ctr < 100)	// recalculate max. 100 times
					{
						theta1ctr++;
						goto prpt;
					}
					else			// terminate photon
					{
						num_theta1++;	// increment the counter for 'theta1' - # photons terminated due to incidence angle > 1.57 or < 0 radian
						flag_abs = 1;
						theta1ctr = 0;
						goto baexit;
					}
				}
				else		// 0< theta1 < 1.57 radians; continue photon transport
					theta1 = acosf(theta1);
		

				// check for conditions where photon can only reflect
				if (flag_call_transmit == 1)	// only valid when photon within the column and can transmit outside the column. asin(n1/n2) -> nan
				{
					if (theta1 > asin(n2/n1))	// critical angle condition for total internal reflection (TIR)
					{
						Pr = 1.0f;		// TIR occurs
						Pt = 0.0f;
					}
			       		else if ( theta1 < epsilon ) 	// theta1 ~= 0, then always reflect
					{
				        	theta1 = 0.00042;       // make theta1 a very small number, to avoid getting nan probabilities
					        theta2 = asinf((float)(n1/n2)*sin(theta1));     // refracted/transmitted angle in radians

					        // Using Fresnel's law, compute probability of reflection and transmission 
					        Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
				        	Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
					else    // the ray will transmit
					{
				        	theta2 = asinf((float)(n1/n2)*sin(theta1));     // refracted/transmitted angle in radians
					        
					        // Using Fresnel's law, compute probability of reflection and transmission 
				        	Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
					        Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}

				}
				else if (flag_call_transmit == 0)	// outside the column
				{		
					if((n1/n2) < 1.57f)	
					{
						if (theta1 > asin(n1/n2))	// critical angle condition for total internal reflection (TIR)
						{
							Pr = 1.0f;		// TIR occurs
							Pt = 0.0f;
						}
					}
					else if ( theta1 < epsilon )	// theta1 ~= 0, then always reflect
					{
						theta1 = 0.00042;	// make theta1 a very small number, to avoid getting nan probabilities
						theta2 = asinf((float)(n1/n2)*sin(theta1)); 	// refracted/transmitted angle in radians
	
						// Using Fresnel's law, compute probability of reflection and transmission 
						Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
						Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}
					else	// photon transmits
					{
						theta2 = asinf((float)(n2/n1)*sin(theta1));

						// Using Fresnel's law, compute probability of reflection and transmission 
						Pr = 1/2.0f*( (pow(tan(theta1-theta2),2)/pow(tan(theta1+theta2),2)) + (pow(sin(theta1-theta2),2)/pow(sin(theta1+theta2),2)) );
						Pt = 1/2.0f*( ((sin(2.0f*theta1)*sin(2.0f*theta2))/(pow(sin(theta1+theta2),2)*pow(cos(theta1-theta2),2))) + ((sin(2.0f*theta1)*sin(2.0f*theta2))/pow(sin(theta1+theta2),2)) );
					}

				}


				// normalize Pr and Pt
				temp_norm = Pr + Pt;
				Pr = Pr/temp_norm;
				Pt = Pt/temp_norm;


				if(ranecu(seed) < Pr)				// reflection
				{
					trans_dir_cos(dcos, normal, theta1, theta2, 0, info);  // compute reflected directional cosines


					// condition to check that reflected vector is within 1.57 radians from original normal
					angle_oldN_R = dot_product(old_normal, dcos);
					angle_oldN_R = acosf(angle_oldN_R);


					if (angle_oldN_R > 1.57f) // > 1.57 radians, reperturb the normal
					{
						reperturb_ctr++;

						if(reperturb_ctr < 4)		// maximum 3 times reperturb
							goto reperturb;
						else				// calculate using smooth surface normal (old_normal)
						{
							normal[0] = old_normal[0];
							normal[1] = old_normal[1];
							normal[2] = old_normal[2];

							dcos[0] = old_dcos[0];
							dcos[1] = old_dcos[1];
							dcos[2] = old_dcos[2];

							dcos_temp[0] = -dcos[0];
							dcos_temp[1] = -dcos[1];
							dcos_temp[2] = -dcos[2];

							reperturb_ctr = 0;

							if(oldN_Rctr < 25)	// max resample 25 times (25*3 reperturb = 75 times)
							{
								oldN_Rctr++;
								goto no_perturbation;
							}
							else			// terminate photon
							{
								num_theta1++;	// increment the counter for 'theta1' - # photons terminated due to incidence angle > 1.57 or < 0 radian
								flag_abs = 1;
								oldN_Rctr = 0;
								goto baexit;
							}
						}
					}

					if (flag_call_transmit == 0)		// reflects between columns, calculate distance using transmit()
					{
						flagCCT = 0;

						trans_flag = transmit(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, d_max, sensorRefl, ydim, flagCCT, h_num_detected_prim);

						if (trans_flag == 1)		// photon terminated
							flag_abs = 1;
						else if (trans_flag == 0)
							goto prpt;				
					}
				}
				else						// transmission
				{
					trans_dir_cos(dcos, normal, theta1, theta2, 1, info);	// compute transmitted directional cosines

					if (flag_call_transmit == 1)		// photon exits current column
					{
						flag_call_transmit = 0;
						flagCCT = 0;

						trans_flag = transmit(pos, dcos, normal, seed, xdetector, ydetector, H, top_absfrac, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, myimage, info, d_max, sensorRefl, ydim, flagCCT, h_num_detected_prim);

						if (trans_flag == 1)		// photon terminated
							flag_abs = 1;
						else if (trans_flag == 0)	// hits a column
							goto prpt;		// check again to see if it gets reflected or transmitted
					}			
				}
			} // else prpt ends
		}	// else ends

	baexit:
	   return flag_abs;

	}	

#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// Photon gets transmitted, calculate the new position where it hits next column or boundary
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
__device__ int transmit(float3 *pos, float3 *dcos, float3 *normal, int2* seed, float xdetector, float ydetector, float H, 
float top_absfrac, float beta, float d_min, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y,
unsigned long long int *myimage, size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float d_max, 
float sensorRefl, int flagCCT, float *Xc, float *Yc, struct histStruct * dPhotonHist, int boolToCollectData)
{
	float d_nextCol = 0.0f;		// distance to next column	
	float d_top = 0.0f;		// distance to top surface
	float d_bottom = 0.0f;		// distance to bottom surface (sensor plane)
	int photon_exit = 0;		// flag - photon terminates (lost/detected/absorbed) (yes=1; no=0)
	int reflbtm = 0;		// flag - photon terminated during call to refl_bottom() (yes=1; no=0)
	int newnormalctr=0;		// counter - if angle between inverted dir. cosine and rough normal > 1.57 or < 0 radians (recalculate normal; max. 100 times)
	int newnormalctr2=0;
	float newangle = 0.0f;
	float cos_newangle = 0.0f;
	float3 temp_pos = {0.0f};
	float3 temp_dcos = {0.0f};
	float rr_rnd = 0.0f, theta_rnd = 0.0f;
	float tmp_deno = 0.0f;
	int iii = 0, jjj = 0;

	temp_pos.x = pos->x;
	temp_pos.y = pos->y;
	temp_pos.z = pos->z;

	temp_dcos.x = -dcos->x;		// inverted directional cosines - used to calculate angle between incident and normal vectors
	temp_dcos.y = -dcos->y;
	temp_dcos.z = -dcos->z;

	/*if(mytid == 6 || mytid == 7)
	{
			printf("tid: %d in transmit\n", mytid);
	}*/
	
	if(flagCCT == 1)	// columnar crosstalk occurs
	{
		// photon moves to adjacent column with no change in its dir. cosines. d_nextcol = 0. adjacent column has random orientation.
		newnormal1:
			normal->x = temp_dcos.x;		// invert incident dir. cosines
			normal->y = temp_dcos.y;
			normal->z = temp_dcos.z;

			RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

			tmp_deno = sqrt(normal->x*normal->x + normal->y*normal->y);

			// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
			normal->z = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
			normal->x = normal->x/tmp_deno;		
			normal->y = normal->y/tmp_deno;

			// perturb the normal according to Beta
			if(beta > 0.0f)
				RoughSurface(normal, seed, beta);

			// find the angle between normal and -dcos
			cos_newangle = dot_product(&temp_dcos, normal);
			newangle = acosf(cos_newangle);

			if ( (newangle < 0.0f) || (newangle > 1.57f) )	// new normal within +- 1.57 radians from inverted dcos
			{						// keep looping until 'newangle' within 1.57 radians (max. 100 iterations)
				if(newnormalctr < 100)
				{
					newnormalctr++;
					goto newnormal1;			
				}
				else 					// else terminate photon
				{
					atomicAdd(&num_theta1,1);	// increment counter - # photons terminated due to incidence angle > 1.57 or < 0 radian
					photon_exit = 1;
					newnormalctr = 0;
					goto exitnow;
				}						
			}
		
			photon_exit = 0;	// photon still alive
	}
	else		// no columnar crosstalk
	{
		// sample distance uniformly between d_min and d_max to next column
		d_nextCol = ranecu(seed) * (d_max - d_min) + d_min;

		// new position of the photon. 
		pos->x = temp_pos.x + dcos->x * d_nextCol;
		pos->y = temp_pos.y + dcos->y * d_nextCol;
		pos->z = temp_pos.z + dcos->z * d_nextCol;
	
		if(mytid < dev_numPhotonHist && boolToCollectData == 1)
		{
			dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
			dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
			dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
			dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
			dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
			dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
			dPhotonHist[mytid].histCounter ++;
		}
		
		/*if(mytid == 6 || mytid == 7)
		{
			printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
		}*/
			
		d_top = ((H/2.0f) - temp_pos.z)/dcos->z;
		d_bottom  = ((-H/2.0f) - temp_pos.z)/dcos->z;

		// new position within detector boundaries? - if false, photon LOST
		if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) )
		{
			atomicAdd(&num_lost, 1);	// increment # photons lost
			
			if(mytid < dev_numPhotonHist && boolToCollectData == 1)
			{
				dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
				dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
				dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
				dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
				dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
				dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 2;
				dPhotonHist[mytid].histCounter ++;
			}
			
			/*if(mytid == 6 || mytid == 7)
			{
				printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
			}*/
			
			photon_exit = 1;
			goto exitnow;
		}

		if ( (pos->z < -H/2.0f) || (pos->z > H/2.0f)  )
		{
			if( (d_top < d_nextCol) && (d_top > epsilon) )	// d_top < d_nextCol: photon reflects from top surface
			{
				pos->x = temp_pos.x + dcos->x * d_top;
				pos->y = temp_pos.y + dcos->y * d_top;
				pos->z = H/2.0f;

				if(mytid < dev_numPhotonHist && boolToCollectData == 1)
				{
					dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
					dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
					dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
					dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
					dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
					dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
					dPhotonHist[mytid].histCounter ++;
				}
				
				/*if(mytid == 6 || mytid == 7)
				{
					printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
				}*/
			
				atomicAdd(&photon_distance, d_top);	// add distance to mean free path of photon
				photon_exit = 0;			
			}
			else if( (d_bottom < d_nextCol) && (d_bottom > epsilon) )	// d_bottom < d_nextCol, photon hits sensor plane and is reflected/detected
			{
				pos->x = temp_pos.x + dcos->x * d_bottom;
				pos->y = temp_pos.y + dcos->y * d_bottom;
				pos->z = -H/2.0f;

				if(mytid < dev_numPhotonHist && boolToCollectData == 1)
				{
					dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
					dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
					dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
					dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
					dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
					dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
					dPhotonHist[mytid].histCounter ++;
				}
				
				/*if(mytid == 6 || mytid == 7)
				{
					printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
				}*/
			
				atomicAdd(&photon_distance, d_bottom);

				// non-ideal sensor - reflects back sensorRefl% of photons into the current column; detects rest
				if(ranecu(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
				{
					photon_exit = 0;

					// normal pointing +z-direction
					normal->x = 0.0f; normal->y = 0.0f; normal->z = 1.0f;

					// obtain reflected dcos from the bottom (specular reflection; 
					// bottom surface is smooth, so normal is not perturbed)
					trans_dir_cos(dcos, normal, 0.0f, 0.0f, 0, mytid, info);	// reflection only, so 'refl_theta,trans_theta' = 0

					// sample new distance and place new column
					reflbtm = refl_bottom(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max, mytid, Xc, Yc, dPhotonHist, boolToCollectData);

					if(reflbtm == 1)	// photon terminated in refl_bottom()
					{
						photon_exit = 1;
						goto exitnow;
					}

					// hits top surface after reflecting back
					if ( (fabs(pos->z - (H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	
					{
						goto mytopsurface;
					}
				}
				else		// does not reflect back into column; detected
				{
					photon_exit = 1;	
					atomicAdd(&num_detect, 1);

					if(mytid < dev_numPhotonHist && boolToCollectData == 1)
					{
						dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
						dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
						dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
						dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
						dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
						dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 1;
						dPhotonHist[mytid].histCounter ++;
					}
					
					/*if(mytid == 6 || mytid == 7)
					{
						printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
					}*/
			
					iii = floor((pos->x-lbound_x)/pixelsize);	// x,y pixel number in the sensor plane of the detector
					jjj = floor((pos->y-lbound_y)/pixelsize);

					// if photon is detected within lower and upper bounds of point response function: accumulate the signal contribution
					if( (pos->x <= ubound_x) && (pos->y <= ubound_y) && (pos->x >= lbound_x) && (pos->y >= lbound_y) )
					{	
						unsigned long long int* current_img = (unsigned long long int*)((char*)myimage + iii * pitch);
						atomicAdd(&current_img[jjj],1);
					}

					atomicAdd(&num_detected_primary[info[mytid].str_histnum-1],1);	// start array from 0.str_histnum starts from 1
				
					goto exitnow;
				}	
			}
			else	// terminate photon
			{
				atomicAdd(&num_lost, 1);

				if(mytid < dev_numPhotonHist && boolToCollectData == 1)
				{
					dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
					dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
					dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
					dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
					dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
					dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 2;
					dPhotonHist[mytid].histCounter ++;
				}
				
				/*if(mytid == 6 || mytid == 7)
				{
					printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
				}*/
			
				photon_exit = 1;
				goto exitnow;
			}
		}
		else
			atomicAdd(&photon_distance, d_nextCol);		// add distance to mean free path


			// sample new normal to determine orientation of new column.
		newnormal:
			normal->x = temp_dcos.x;		// invert dcos of incident vector
			normal->y = temp_dcos.y;
			normal->z = temp_dcos.z;

			RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

			tmp_deno = sqrt(normal->x*normal->x + normal->y*normal->y);

			// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
			normal->z = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
			normal->x = normal->x/tmp_deno;		
			normal->y = normal->y/tmp_deno;

			// perturb the normal according to Beta
			if(beta > 0.0f)
				RoughSurface(normal, seed, beta);

			// angle between normal and -dcos
			cos_newangle = dot_product(&temp_dcos, normal);
			newangle = acosf(cos_newangle);

			if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 1.57 radians from inverted dcos
			{						// keep looping until 'newangle' within 1.57 radians (max. 100 times)
				if(newnormalctr < 100)
				{
					newnormalctr++;
					goto newnormal;			
				}
				else // kill it
				{
					atomicAdd(&num_theta1,1);
					photon_exit = 1;
					newnormalctr = 0;
					goto exitnow;
				}
						
			}
	
			// check if the photon enters another column or get lost (hit detector side)/ reflected (at top surface)/ detected (at bottom surface)
	
			// hits side of detector?
			if ( (fabs(pos->x-0.0f) < epsilon) || (fabs(pos->x-xdetector) < epsilon) || (fabs(pos->y-0.0f) < epsilon) || (fabs(pos->y-ydetector) < epsilon) )		
			{
				atomicAdd(&num_lost, 1);

				if(mytid < dev_numPhotonHist && boolToCollectData == 1)
				{
					dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
					dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
					dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
					dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
					dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
					dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 2;
					dPhotonHist[mytid].histCounter ++;
				}
				
				/*if(mytid == 6 || mytid == 7)
				{
					printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
				}*/
			
				photon_exit = 1;
				goto exitnow;

			}
	

			// hits top surface?
		     mytopsurface:

			if ( (fabs(pos->z - (H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	// gets reflected or absorbed
			{
				normal->x = 0.0f;
				normal->y = 0.0f;
				normal->z = -1.0f;

				// top surface absorption - using absorption coefficient 'top_absfrac'
				if ( (top_absfrac > 0.0f) && (ranecu(seed) < top_absfrac) )	// photon absorbed
				{
					atomicAdd(&num_abs_top, 1);	// increment # photons absorbed at top counter
	
					if(mytid < dev_numPhotonHist && boolToCollectData == 1)
					{
						dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
						dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
						dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
						dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
						dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
						dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 3;
						dPhotonHist[mytid].histCounter ++;
					}
					
					/*if(mytid == 6 || mytid == 7)
					{
						printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
					}*/
			
					photon_exit = 1;
					goto exitnow;
				}
				else	// not absorbed at top
				{
					// assign new directional cosines (isotropic surface)
					dcos->z = -fabs((ranecu(seed) * 2.0f) - 1.0f);
					rr_rnd = sqrt(1.0f - dcos->z*dcos->z);
					theta_rnd = ranecu(seed)*twopipen;	
	
					dcos->x=rr_rnd*cos(theta_rnd);
					dcos->y=rr_rnd*sin(theta_rnd);

					temp_pos.x = pos->x;
					temp_pos.y = pos->y;
					temp_pos.z = pos->z;

					temp_dcos.x = -dcos->x;
					temp_dcos.y = -dcos->y;
					temp_dcos.z = -dcos->z;

					// sample distance uniformly between d_min and d_max to next column
					d_nextCol = ranecu(seed) * (d_max - d_min) + d_min;

					// distance to bottom surface: if d_bottom < d_nextCol, photon is detected.
					d_bottom  = ((-H/2.0f) - temp_pos.z)/dcos->z;

					// new position of the photon. 
					pos->x = temp_pos.x + dcos->x * d_nextCol;
					pos->y = temp_pos.y + dcos->y * d_nextCol;
					pos->z = temp_pos.z + dcos->z * d_nextCol;
					
					if(mytid < dev_numPhotonHist && boolToCollectData == 1)
					{
						dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
						dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
						dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
						dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
						dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
						dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
						dPhotonHist[mytid].histCounter ++;
					}
					
					/*if(mytid == 6 || mytid == 7)
					{
						printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
					}*/
			
					// pos is within detector boundaries? - if false, photon lost
					if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) )
					{
						atomicAdd(&num_lost, 1);	// increment # photons lost
	
						if(mytid < dev_numPhotonHist && boolToCollectData == 1)
						{
							dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
							dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
							dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
							dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
							dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
							dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 2;
							dPhotonHist[mytid].histCounter ++;
						}
						
						/*if(mytid == 6 || mytid == 7)
						{
							printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
						}*/
			
						photon_exit = 1;
						goto exitnow;
					}

					if ( (pos->z < -H/2.0f) || (pos->z > H/2.0f)  )
					{
						if( (d_bottom < d_nextCol) && (d_bottom > epsilon) ) // dist. to bottom < dist. to next column
						{
							pos->x = temp_pos.x + dcos->x * d_bottom;
							pos->y = temp_pos.y + dcos->y * d_bottom;
							pos->z = -H/2.0f;
			
							if(mytid < dev_numPhotonHist && boolToCollectData == 1)
							{
								dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
								dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
								dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
								dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
								dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
								dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 0;
								dPhotonHist[mytid].histCounter ++;
							}
							
							/*if(mytid == 6 || mytid == 7)
							{
								printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
							}*/
			
							atomicAdd(&photon_distance, d_bottom);

							// non-ideal sensor - reflects back sensorRefl% of photons into the current column; detects rest
							if(ranecu(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
							{
								photon_exit = 0;

								// normal pointing (0,0,1)
								normal->x = 0.0f; normal->y = 0.0f; normal->z = 1.0f;

								// obtain reflected dcos from the bottom (specular reflection; 
								// bottom surface is smooth, do not perturb the normal)
								trans_dir_cos(dcos, normal, 0.0f, 0.0f, 0, mytid, info);   // reflection only so 'refl_theta,trans_theta' = 0

								// sample new distance and place new column there
								reflbtm = refl_bottom(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max, mytid, Xc, Yc, dPhotonHist, boolToCollectData);

								if(reflbtm == 1)
								{
									photon_exit = 1;
									goto exitnow;
								}

								// hits top surface after reflecting back
								if ( (fabs(pos->z - (H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	
								{
									goto mytopsurface;
								}
							}
							else	// not reflected back into column; detected
							{
								photon_exit = 1;
								atomicAdd(&num_detect, 1);

								if(mytid < dev_numPhotonHist && boolToCollectData == 1)
								{
									dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
									dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
									dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
									dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
									dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
									dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 1;
									dPhotonHist[mytid].histCounter ++;
								}
								
								/*if(mytid == 6 || mytid == 7)
								{
									printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
								}*/
				
								iii = floor((pos->x-lbound_x)/pixelsize);	// x,y pixel number
								jjj = floor((pos->y-lbound_y)/pixelsize);

								// if photon gets detected within lower and upper bounds: accumulate signal contribution
								if( (pos->x <= ubound_x) && (pos->y <= ubound_y) && (pos->x >= lbound_x) && (pos->y >= lbound_y) )
								{	
									unsigned long long int* current_img = (unsigned long long int*)((char*)myimage + iii * pitch);
									atomicAdd(&current_img[jjj],1);
								}

								atomicAdd(&num_detected_primary[info[mytid].str_histnum-1],1);// increment # detected per primary

								goto exitnow;	
							}	
						}
						else	// terminate photon
						{
							atomicAdd(&num_lost, 1);

							if(mytid < dev_numPhotonHist && boolToCollectData == 1)
							{
								dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
								dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
								dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
								dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
								dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
								dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 2;
								dPhotonHist[mytid].histCounter ++;
							}
							
							/*if(mytid == 6 || mytid == 7)
							{
								printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
							}*/
			
							photon_exit = 1;
							goto exitnow;
						}
					}
					else	// photon lies between top and bottom surfaces of detector
						atomicAdd(&photon_distance, d_nextCol);		// add distance travelled to global variable

					// sample new normal to determine orientation of new column.
			  newnormal_TOP:
					normal->x = temp_dcos.x;		// invert dcos of incident vector
					normal->y = temp_dcos.y;
					normal->z = temp_dcos.z;

					RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

					tmp_deno = sqrt(normal->x*normal->x + normal->y*normal->y);

					// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
					normal->z = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
					normal->x = normal->x/tmp_deno;		
					normal->y = normal->y/tmp_deno;

					// perturb the normal according to beta
					if(beta > 0.0f)
						RoughSurface(normal, seed, beta);

					// angle between normal and -dcos
					cos_newangle = dot_product(&temp_dcos, normal);
					newangle = acosf(cos_newangle);

					if ( (newangle < 0.0f) || (newangle > 1.57f) )	// new normal within +- 1.57 radians from inverted dcos
					{						// keep looping until 'newangle' within 1.57 radians (max. 100 times)
						if(newnormalctr2 < 100)			
						{
							newnormalctr2++;
							goto newnormal_TOP;					
						}
						else 					// terminate photon
						{
							atomicAdd(&num_theta1,1);
							photon_exit = 1;
							newnormalctr2 = 0;
							goto exitnow;
						}
					}
					photon_exit = 0;
				}
			}	// hit top ends
	

			// hit bottom? 
			if ( fabs(pos->z - (-H/2.0f)) < epsilon )	// gets detected
			{
				// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
				if(ranecu(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
				{
					photon_exit = 0;

					// normal pointing (0,0,1)
					normal->x = 0.0f; normal->y = 0.0f; normal->z = 1.0f;

					// obtain reflected dcos from the bottom (specular reflection; 
					// bottom surface is smooth, do not perturb the normal)
					trans_dir_cos(dcos, normal, 0.0f, 0.0f, 0, mytid, info);	// reflection only so 'refl_theta,trans_theta' = 0

					// sample new distance and place new column
					reflbtm = refl_bottom(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max, mytid, Xc, Yc, dPhotonHist, boolToCollectData);

					if(reflbtm == 1)
					{
						photon_exit = 1;
						goto exitnow;
					}

					// hits top surface after reflecting back?
					if ( (fabs(pos->z - (H/2.0f)) < epsilon) && (dcos->z > 0.0f) )	
					{
						goto mytopsurface;
					}
				}
				else	// detected
				{
					atomicAdd(&num_detect, 1);

					if(mytid < dev_numPhotonHist && boolToCollectData == 1)
					{
						dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
						dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
						dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
						dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
						dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
						dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 1;
						dPhotonHist[mytid].histCounter ++;
					}
					
					/*if(mytid == 6 || mytid == 7)
					{
						printf("transmit - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
					}*/
			
					photon_exit = 1;

					iii = floor((pos->x-lbound_x)/pixelsize);	// x,y pixel number in the sensor plane of the detector
					jjj = floor((pos->y-lbound_y)/pixelsize);

					// if the photon gets detected within lower and upper bounds: accumulate the signal contribution
					if( (pos->x <= ubound_x) && (pos->y <= ubound_y) && (pos->x >= lbound_x) && (pos->y >= lbound_y) )
					 {	
						unsigned long long int* current_img = (unsigned long long int*)((char*)myimage + iii * pitch);
						atomicAdd(&current_img[jjj],1);
					 }

					atomicAdd(&num_detected_primary[info[mytid].str_histnum-1],1);	// increment # detected per primary
	
					goto exitnow;
				}
			}
		} // else flagCCT ends
	
	exitnow:
	 return photon_exit;	
	}
	
#else	// C code

	int transmit(float *pos, float *dcos, float *normal, int* seed, float xdetector, float ydetector, float H, float top_absfrac, float beta, float d_min, int pixelsize, 
	float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, float d_max, float sensorRefl, int ydim, int flagCCT, 		int *h_num_detected_prim)
	{
		float d_nextCol = 0.0f;		// distance to next column	
		float d_top = 0.0f;		// distance to top surface
		float d_bottom = 0.0f;		// distance to bottom surface
		int photon_exit = 0;		// flag - photon terminates (lost/detected/absorbed) (yes=1; no=0)
		int reflbtm = 0;		// flag - photon terminated during call to refl_bottom() (yes=1; no=0)
		int newnormalctr = 0;		// counter - if angle between inverted dir. cosine and rough normal > 1.57 or < 0 radians (recalculate normal; max. 100 times)
		int newnormalctr2 = 0;
		float newangle = 0.0f;
		float cos_newangle = 0.0f;
		float temp_pos[3] = {0.0f};
		float temp_dcos[3] = {0.0f};
		float rr_rnd = 0.0f, theta_rnd = 0.0f;
		float tmp_deno = 0.0f;
		int iii = 0, jjj = 0;


		temp_pos[0] = pos[0];
		temp_pos[1] = pos[1];
		temp_pos[2] = pos[2];

		temp_dcos[0] = -dcos[0];	// inverted directional cosine - used to calculate angle between incident vector and normal
		temp_dcos[1] = -dcos[1];
		temp_dcos[2] = -dcos[2];

		if(flagCCT == 1)	// columnar crosstalk occurs
		{
			// cross over to adjacent column. no change in dir. cosines. d_nextcol = 0. adjacent column has random orientation.
			newnormal1:
				normal[0] = temp_dcos[0];		// invert dcos of incident vector
				normal[1] = temp_dcos[1];
				normal[2] = temp_dcos[2];

				RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

				tmp_deno = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

				// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
				normal[2] = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
				normal[0] = normal[0]/tmp_deno;		
				normal[1] = normal[1]/tmp_deno;

				// perturb the normal according to Beta
				if(beta > 0.0f)
					RoughSurface(normal, seed, beta);

				// find the angle between Normal and -Dcos
				cos_newangle = dot_product(temp_dcos, normal);
				newangle = acosf(cos_newangle);

				if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 1.57 radians from inverted dcos
				 {						// keep looping until 'newangle' within 1.57 radians (max. 100 times)
					if(newnormalctr < 100)
					{
						newnormalctr++;
						goto newnormal1;			
					}
					else 					// terminate photon
					{
						num_theta1++;
						photon_exit = 1;
						newnormalctr = 0;
						goto exitnow;
					}
						
				 }

				photon_exit = 0;	// photon still alive
		}
		else		// no columnar crosstalk occurs
		{

			// sample distance uniformly between d_min and d_max to next column
			d_nextCol = ranecu(seed) * (d_max - d_min) + d_min;

			// new position of the photon. 
			pos[0] = temp_pos[0] + dcos[0] * d_nextCol;
			pos[1] = temp_pos[1] + dcos[1] * d_nextCol;
			pos[2] = temp_pos[2] + dcos[2] * d_nextCol;

			d_top = ((H/2.0f) - temp_pos[2])/dcos[2];
			d_bottom  = ((-H/2.0f) - temp_pos[2])/dcos[2];

			// condition to check that pos is within detector boundaries - if true, photon LOST
			if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) )
			{
				num_lost++;
				photon_exit = 1;
				goto exitnow;
			}

			if ( (pos[2] < -H/2.0f) || (pos[2] > H/2.0f)  )
				{
					if( (d_top < d_nextCol) && (d_top > epsilon) )	// d_top < d_nextCol: photon reflects from the top surface
					{
						pos[0] = temp_pos[0] + dcos[0] * d_top;
						pos[1] = temp_pos[1] + dcos[1] * d_top;
						pos[2] = H/2.0f;
				
						photon_distance = photon_distance + d_top;
						photon_exit = 0;			
					}
					else if( (d_bottom < d_nextCol) && (d_bottom > epsilon) )	// d_bottom < d_nextCol: photon detected.
					{
						pos[0] = temp_pos[0] + dcos[0] * d_bottom;
						pos[1] = temp_pos[1] + dcos[1] * d_bottom;
						pos[2] = -H/2.0f;

						photon_distance = photon_distance + d_bottom;

						// non-ideal sensor - reflects back sensorRefl% of photons into the current column; detects rest
						if(ranecu(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
						{
		
							photon_exit = 0;

							// normal pointing +z-direction
							normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 1.0f;

							// obtain reflected dcos from the bottom (specular reflection; 
							// bottom surface is smooth, do not perturb the normal)
							trans_dir_cos(dcos, normal, 0.0f, 0.0f, 0, info);	// reflection only so 'refl_theta,trans_theta' = 0

							// sample new distance and place new column there
							reflbtm = refl_bottom(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

							if(reflbtm == 1)
							{
								photon_exit = 1;	// photon terminated in refl_bottom()
								goto exitnow;
							}


							// hits top surface after reflecting back?
							if ( (fabs(pos[2] - (H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	
							{
								goto mytopsurface;
							}

						}
						else	// not reflected back into column; photon detected
						{
							photon_exit = 1;	
							num_detect++;

							iii = floor((pos[0]-lbound_x)/pixelsize);	// x,y pixel number in the sensor plane of the detector
							jjj = floor((pos[1]-lbound_y)/pixelsize);

							// if the photon gets detected within lower and upper bounds: accumulate the signal contribution
							if( (pos[0] <= ubound_x) && (pos[1] <= ubound_y) && (pos[0] >= lbound_x) && (pos[1] >= lbound_y) )
							 {	
								outputimage_.newimageopt[iii][jjj]++;
							 }

							h_num_detected_prim[info.str_histnum]++;	// increment # detected per primary
				
							goto exitnow;
						}	
					}
					else	// terminate photon
					{
						num_lost++;
						photon_exit = 1;
						goto exitnow;
					}
				}
			else		// photon lies between top and bottom surfaces of detector
				photon_distance = photon_distance + d_nextCol;		// add distance to mean free path of photon


			// sample new normal to determine orientation of new column.
		newnormal:
			normal[0] = temp_dcos[0];		// invert dcos of incident vector
			normal[1] = temp_dcos[1];
			normal[2] = temp_dcos[2];

			RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

			tmp_deno = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

			// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
			normal[2] = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
			normal[0] = normal[0]/tmp_deno;		
			normal[1] = normal[1]/tmp_deno;

			// perturb the normal according to Beta
			if(beta > 0.0f)
				RoughSurface(normal, seed, beta);

			// angle between normal and -dcos
			cos_newangle = dot_product(temp_dcos, normal);
			newangle = acosf(cos_newangle);

			if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 1.57 radians from inverted dcos
			 {						// keep looping until 'newangle' within 1.57 radians (max. 100 iterations)
				if(newnormalctr < 100)
				{
					newnormalctr++;
					goto newnormal;			
				}
				else 					// terminate photon
				{
					num_theta1++;
					photon_exit = 1;
					newnormalctr = 0;
					goto exitnow;
				}
						
			 }
	
			// check if the photon enters another column or gets lost (hit detector side)/ reflected (detector top)/ detected (detector bottom)
	
			// hits side of detector?
			if ( (fabs(pos[0]-0.0f) < epsilon) || (fabs(pos[0]-xdetector) < epsilon) || (fabs(pos[1]-0.0f) < epsilon) || (fabs(pos[1]-ydetector) < epsilon) )		
			{
				num_lost++;
				photon_exit = 1;
				goto exitnow;

			}


			// hit top?
		     mytopsurface:

			if ( (fabs(pos[2] - (H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	// gets reflected or absorbed
			{
				normal[0] = 0.0f;
				normal[1] = 0.0f;
				normal[2] = -1.0f;

				if ( (top_absfrac > 0.0f) && (ranecu(seed) < top_absfrac) )	// photon gets absorbed at top surface
				{
					num_abs_top++;
					photon_exit = 1;
					goto exitnow;
				}
				else	// photon reflected from top
				{
					// assign new directional cosines (isotropic top surface)
					dcos[2] = -fabs((ranecu(seed) * 2.0f) - 1.0f);
					rr_rnd = sqrt(1.0f - dcos[2]*dcos[2]);
					theta_rnd = ranecu(seed)*twopipen;	
	
					dcos[0]=rr_rnd*cos(theta_rnd);
					dcos[1]=rr_rnd*sin(theta_rnd);

					temp_pos[0] = pos[0];
					temp_pos[1] = pos[1];
					temp_pos[2] = pos[2];

					temp_dcos[0] = -dcos[0];
					temp_dcos[1] = -dcos[1];
					temp_dcos[2] = -dcos[2];

					// sample distance uniformly between d_min and d_max to next column
					d_nextCol = ranecu(seed) * (d_max - d_min) + d_min;

					// distance to bottom surface: if d_bottom < d_nextCol, photon detected.
					d_bottom  = ((-H/2.0f) - temp_pos[2])/dcos[2];

					// compute the new position of the photon. 
					pos[0] = temp_pos[0] + dcos[0] * d_nextCol;
					pos[1] = temp_pos[1] + dcos[1] * d_nextCol;
					pos[2] = temp_pos[2] + dcos[2] * d_nextCol;

					// check new position is within detector boundaries? - if false, photon lost
					if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) )
					{
						num_lost++;
						photon_exit = 1;
						goto exitnow;
					}

					if ( (pos[2] < -H/2.0f) || (pos[2] > H/2.0f)  )
						{
							if( (d_bottom < d_nextCol) && (d_bottom > epsilon) )
							{
								pos[0] = temp_pos[0] + dcos[0] * d_bottom;
								pos[1] = temp_pos[1] + dcos[1] * d_bottom;
								pos[2] = -H/2.0f;

								photon_distance = photon_distance + d_bottom;

								// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
								if(ranecu(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
								{

									photon_exit = 0;		
			
									// normal pointing (0,0,1)
									normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 1.0f;

									// obtain reflected dcos from the bottom (specular reflection; 
									// bottom surface is smooth, do not perturb the normal)
									trans_dir_cos(dcos, normal, 0.0f, 0.0f, 0, info);	// reflection only so 'refl_theta,trans_theta' = 0

									// sample new distance and place new column
									reflbtm = refl_bottom(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

									if(reflbtm == 1)
									{
										photon_exit = 1;
										goto exitnow;
									}

									// hits top surface after reflecting back?
									if ( (fabs(pos[2] - (H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	
									{
										goto mytopsurface;
									}

								}
								else		// not reflected back into column; detected
								{
										photon_exit = 1;
										num_detect++;

										iii = floor((pos[0]-lbound_x)/pixelsize);	// x,y pixel number
										jjj = floor((pos[1]-lbound_y)/pixelsize);

										// photon gets detected within lower and upper bounds: accumulate signal contribution
										if( (pos[0] <= ubound_x) && (pos[1] <= ubound_y) && (pos[0] >= lbound_x) && (pos[1] >= lbound_y) )
										 {	
											outputimage_.newimageopt[iii][jjj]++;
										 }

										h_num_detected_prim[info.str_histnum]++;	// increment # detected per primary

										goto exitnow;	
								}	
							}
							else	// terminate photon
							{
								num_lost++;
								photon_exit = 1;
								goto exitnow;
							}
						}
					else
						photon_distance = photon_distance + d_nextCol;		// add distance to mean free path of photon

					// sample new normal to determine orientation of new column.
			  newnormal_TOP:
					normal[0] = temp_dcos[0];		// invert dcos of incident vector
					normal[1] = temp_dcos[1];
					normal[2] = temp_dcos[2];

					RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

					tmp_deno = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

					// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
					normal[2] = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
					normal[0] = normal[0]/tmp_deno;		
					normal[1] = normal[1]/tmp_deno;

					// perturb the normal according to Beta
					if(beta > 0.0f)
						RoughSurface(normal, seed, beta);

					// angle between normal and -dcos
					cos_newangle = dot_product(temp_dcos, normal);
					newangle = acosf(cos_newangle);

					if ( (newangle < 0.0f) || (newangle > 1.57f) )	// new normal within +- 1.57 radians from inverted dcos
					 {						// keep looping until 'newangle' within 1.57 radians (max. 100 iterations)
						if(newnormalctr2 < 100)	
						{
							newnormalctr2++;
							goto newnormal_TOP;			
						}
						else 					// terminate photon
						{
							num_theta1++;
							photon_exit = 1;
							newnormalctr2 = 0;
							goto exitnow;
						}
				
					 }
		
					photon_exit = 0;	// photon still alive
				}
			}	// hit top ends
	

			// hit bottom? 
			if ( fabs(pos[2] - (-H/2.0f)) < epsilon )	// gets detected
			{
				// non-ideal sensor - reflects back sensorRefl% of photons into the current column; absorbs rest
				if(ranecu(seed) < sensorRefl)	// reflect back - specular (mirror) reflection
				{
					photon_exit = 0;		
	
					// normal pointing (0,0,1)
					normal[0] = 0.0f; normal[1] = 0.0f; normal[2] = 1.0f;

					// obtain reflected dcos from the bottom (specular reflection; 
					// bottom surface is smooth, do not perturb the normal)
					trans_dir_cos(dcos, normal, 0.0f, 0.0f, 0, info);	// reflection only so 'refl_theta,trans_theta' = 0

					// sample new distance and place new column
					reflbtm = refl_bottom(pos, dcos, normal, xdetector, ydetector, seed, beta, d_min, H, d_max);

					if(reflbtm == 1)
					{
						photon_exit = 1;
						goto exitnow;
					}

					// hits top surface after reflecting back?
					if ( (fabs(pos[2] - (H/2.0f)) < epsilon) && (dcos[2] > 0.0f) )	
					{
						goto mytopsurface;
					}

				}
				else		// detected
				{
					num_detect++;
					photon_exit = 1;

					iii = floor((pos[0]-lbound_x)/pixelsize);	// x,y pixel number in sensor plane of detector
					jjj = floor((pos[1]-lbound_y)/pixelsize);

					// photon gets detected within lower and upper bounds: accumulate the signal contribution
					if( (pos[0] <= ubound_x) && (pos[1] <= ubound_y) && (pos[0] >= lbound_x) && (pos[1] >= lbound_y) )
					 {	
						outputimage_.newimageopt[iii][jjj]++;
					 }

					h_num_detected_prim[info.str_histnum]++;	// increment # detected per primary
	
					goto exitnow;
				}
			}

		} // else flagCCT ends

	exitnow:
	 return photon_exit;	
	}
	
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// photon reflects from sensor_plane or bottom surface, when in between columns. 
// Obtains the next column where it hits.
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
__device__ int refl_bottom(float3 *pos, float3 *dcos, float3 *normal, float xdetector, float ydetector, int2* seed, 
float beta, float d_min, float H, float d_max, int mytid, float *Xc, float *Yc, struct histStruct * dPhotonHist, int boolToCollectData)
{
	float d_nextCol=0.0f;		// distance to next column
	float d_top=0.0f;		// distance to top surface of detector
	int pexit=0;			// flag - photon terminated in this function call (exited: yes=1; no=0)
	int newnormalctr=0;		// counter - angle between inverted dir. cosine and rough normal > 1.57 or < 0 radians (recalculate max. 100 times)
	float tmp_deno=0.0f, cos_newangle=0.0f, newangle=0.0f;
	float3 temp_pos, temp_dcos;

	temp_pos.x = pos->x;
	temp_pos.y = pos->y;
	temp_pos.z = pos->z;

	temp_dcos.x = -dcos->x;		// inverted directional cosines - used to calculate angle between incident and normal vectors
	temp_dcos.y = -dcos->y;
	temp_dcos.z = -dcos->z;

	// sample distance uniformly between d_min and d_max to next column
	d_nextCol = ranecu(seed) * (d_max - d_min) + d_min;

	// distance to top surface
	d_top  = ((H/2.0f) - temp_pos.z)/dcos->z;

	// new position of the photon - specular reflection 
	pos->x = temp_pos.x + dcos->x * d_nextCol;
	pos->y = temp_pos.y + dcos->y * d_nextCol;
	pos->z = temp_pos.z + dcos->z * d_nextCol;

	/*if(mytid == 6 || mytid == 7)
	{
			printf("tid: %d in refl_bottom\n", mytid);
	}*/
	
	// is new position is within detector boundaries? if false, photon lost
	if ( (pos->x < epsilon) || (pos->x > xdetector) || (pos->y < epsilon) || (pos->y > ydetector) )
	{
		atomicAdd(&num_lost, 1);	// increment # photons lost

		if(mytid < dev_numPhotonHist && boolToCollectData == 1)
		{
			dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
			dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
			dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
			dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
			dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
			dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 2;
			dPhotonHist[mytid].histCounter ++;
		}
		/*if(mytid == 6 || mytid == 7)
		{
			printf("refl_bottom - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
		}*/
			
		pexit = 1;			// set the flag
		goto myexit;
	}

	if ( (pos->z > H/2.0f)  )		// if photon's new z-position is above the top surface of detector
	{
		if( (d_top < d_nextCol) && (d_top > epsilon) )		// if distance to top < dist. to next column - photon will hit top surface
		{
			pos->x = temp_pos.x + dcos->x * d_top;
			pos->y = temp_pos.y + dcos->y * d_top;
			pos->z = H/2.0f;
				
			atomicAdd(&photon_distance, d_top);		// add this distance to mean free path of photon
			pexit = 0;			
		}
		else			// terminate photon
		{
			atomicAdd(&num_lost, 1);

			if(mytid < dev_numPhotonHist && boolToCollectData == 1)
			{
				dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
				dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
				dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
				dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
				dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
				dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 2;
				dPhotonHist[mytid].histCounter ++;
			}
			
			/*if(mytid == 6 || mytid == 7)
			{
				printf("refl_bottom - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
			}*/
			
			pexit = 1;
			goto myexit;
		}
	}
	else					// photon z coordinate is less than H/2
	{
		atomicAdd(&photon_distance, d_nextCol);		// add distance to mean free path of photon

		// sample new normal to determine orientation of new column.
		 mynewnormal:
			normal->x = temp_dcos.x;		// invert incident directional cosines
			normal->y = temp_dcos.y;
			normal->z = temp_dcos.z;

			RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

			tmp_deno = sqrt(normal->x*normal->x + normal->y*normal->y);

			// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
			normal->z = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
			normal->x = normal->x/tmp_deno;		
			normal->y = normal->y/tmp_deno;

			// perturb the normal according to Beta
			RoughSurface(normal, seed, beta);

			// angle between normal and -dcos
			cos_newangle = dot_product(&temp_dcos, normal);
			newangle = acosf(cos_newangle);

			if ( (newangle < 0.0f) || (newangle > 1.57f) )	// new rough normal is within +- 1.57 radians from inverted dcos
			{
				if(newnormalctr < 100)	// max 100 times
				{
					newnormalctr++;
					goto mynewnormal;		// keep looping until 'newangle' within 1.57 radians	
				}
				else // kill it
				{
					atomicAdd(&num_theta1,1);   // increment the counter for 'theta1' - # photons terminated due to incidence angle > 1.57 or < 0 radian
					pexit = 1;
					newnormalctr = 0;
					goto myexit;
				}
			}
			pexit = 0;	// photon still alive
	}
	myexit:
		return pexit;
}	
#else	// C code

	int refl_bottom(float *pos, float *dcos, float *normal, float xdetector, float ydetector, int* seed, float beta, float d_min, float H, float d_max)
	{
		float d_nextCol=0.0f;	// distane to next column
		float d_top=0.0f;	// distance to top surface of detector
		int pexit=0;		// flag - photon ternimated in this function call (exited: yes=1; no=0)
		int newnormalctr = 0;	// counter - angle between inverted dir. cosine and rough normal > 1.57 or < 0 radians (recalculate max. 100 times)
		float temp_pos[3], temp_dcos[3];
		float tmp_deno=0.0f, cos_newangle=0.0f, newangle=0.0f;


		temp_pos[0] = pos[0];
		temp_pos[1] = pos[1];
		temp_pos[2] = pos[2];

		temp_dcos[0] = -dcos[0];		// inverted directional cosines - used to calculate angle between incident and normal vectors
		temp_dcos[1] = -dcos[1];
		temp_dcos[2] = -dcos[2];

		// sample distance uniformly between d_min and d_max to next column
		d_nextCol = ranecu(seed) * (d_max - d_min) + d_min;

		// distance to top surface
		d_top  = ((H/2.0f) - temp_pos[2])/dcos[2];

		// compute the new position of the photon. 
		pos[0] = temp_pos[0] + dcos[0] * d_nextCol;
		pos[1] = temp_pos[1] + dcos[1] * d_nextCol;
		pos[2] = temp_pos[2] + dcos[2] * d_nextCol;

		// is new position is within detector boundaries? if false, photon lost
		if ( (pos[0] < epsilon) || (pos[0] > xdetector) || (pos[1] < epsilon) || (pos[1] > ydetector) )
		{
			num_lost++;	// increment # photons lost
			pexit = 1;
			goto myexit;
		}

		if ( (pos[2] > H/2.0f)  )		// photon's new z position is above top surface
		{
				if( (d_top < d_nextCol) && (d_top > epsilon) )		// if distance to top < dist. to next column; photon will hit top surface
				{
					pos[0] = temp_pos[0] + dcos[0] * d_top;
					pos[1] = temp_pos[1] + dcos[1] * d_top;
					pos[2] = H/2.0f;
				
					photon_distance = photon_distance + d_top;	// add this distance to mean free path of photon
					pexit = 0;			
				}
				else			// terminate photon
				{
					num_lost++;
					pexit = 1;
					goto myexit;
				}
		}
		else
		{
			photon_distance = photon_distance + d_nextCol;		// add distance to mean free path of photon

			// sample new normal to determine orientation of new column.
		  	mynewnormal:

				normal[0] = temp_dcos[0];		// invert incident directional cosines
				normal[1] = temp_dcos[1];
				normal[2] = temp_dcos[2];

				RoughSurface(normal, seed, 1.0f);	// beta = 1.0 to get new normal within +-1.57 radians of inverted dcos.

				tmp_deno = sqrt(normal[0]*normal[0] + normal[1]*normal[1]);

				// make z component of normal equal to 0.0f and renormalize (because we want to rotate the normal only in the x-y plane)
				normal[2] = 0.0f;			// normal_z of a cylinder is zero (no tilt assumed)
				normal[0] = normal[0]/tmp_deno;		
				normal[1] = normal[1]/tmp_deno;

				// perturb the normal according to Beta
				RoughSurface(normal, seed, beta);

				// angle between Normal and -Dcos
				cos_newangle = dot_product(temp_dcos, normal);
				newangle = acosf(cos_newangle);

				if ( (newangle < 0.0f) || (newangle > 1.57f) )	// check if new normal is within +- 1.57 radians from inverted dcos
				 {
					if(newnormalctr < 100)	// max 100 times
					{
						newnormalctr++;
						goto mynewnormal;			// keep looping until 'newangle' within 1.57 radians	
					}
					else 			// terminate photon
					{
						num_theta1++;	// increment the counter for 'theta1' - # photons terminated due to incidence angle > 1.57 or < 0 radian
						pexit = 1;
						newnormalctr = 0;
						goto myexit;
					}		
				 }
				pexit = 0;	// photon still alive
		}

	myexit:
	 return pexit;
	}
	
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// calculate dot product of two vectors
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA

	__device__ inline float dot_product(float3 *aa, float3 *b)
	{
		float result = 0.0f;

		result = aa->x*b->x + aa->y*b->y + aa->z*b->z;

	  return result;
	}

#else	// C code

	float dot_product(float *aa, float *b)
	{
		float result = 0.0f;

		result = aa[0]*b[0] + aa[1]*b[1] + aa[2]*b[2];

	  return result;
	}
	
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// compute directional cosines of transmitted vector
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA

	__device__ inline void trans_dir_cos(float3 *dcos, float3 *normal, float refl_theta, float trans_theta, int flag_ref, int mytid, struct start_info *info)
	{
		float cos_angle = 0.0f;
		float norm = 0.0f;
		float3 dcos_temp = {0.0f};

		dcos_temp.x = -dcos->x;		// inverted directional cosines - used to calculate angle between incident and normal vectors
		dcos_temp.y = -dcos->y;
		dcos_temp.z = -dcos->z;
	
		cos_angle = dot_product(&dcos_temp,normal);	// cosine of angle between inverted incident dir. cosine and normal

		if (flag_ref == 0)				// reflection
		{
			dcos->x = 2.0f*cos_angle*normal->x + dcos->x;  // specular reflection
			dcos->y = 2.0f*cos_angle*normal->y + dcos->y;
			dcos->z = 2.0f*cos_angle*normal->z + dcos->z;
		}
		else if (flag_ref == 1)				// transmission	
		{
			 dcos->x= -normal->x*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos->x+(cos_angle*normal->x));
			 dcos->y= -normal->y*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos->y+(cos_angle*normal->y));
			 dcos->z= -normal->z*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos->z+(cos_angle*normal->z));
		}

		// normalize
		norm = sqrt(dcos->x*dcos->x + dcos->y*dcos->y + dcos->z*dcos->z);

		if ((norm < (1.0f - epsilon)) || (norm > (1.0f + epsilon)))
		 {
			dcos->x = dcos->x/norm;
			dcos->y = dcos->y/norm;
			dcos->z = dcos->z/norm;
		 } 

	return;	
	}

#else	// C code

	void trans_dir_cos(float *dcos, float *normal, float refl_theta, float trans_theta, int flag_ref, struct start_info info)
	{
		float cos_angle = 0.0f;
		float norm = 0.0f;
		float dcos_temp[3] = {0.0f};

		dcos_temp[0] = -dcos[0];	// inverted directional cosines - used to calculate angle between incident and normal vectors
		dcos_temp[1] = -dcos[1];
		dcos_temp[2] = -dcos[2];
	
		cos_angle = dot_product(dcos_temp,normal);	// cosine of angle between inverted incident dir. cosine and normal

		if (flag_ref == 0)				// reflection
		{
				dcos[0] = 2.0f*cos_angle*normal[0] + dcos[0];  // specular reflection
				dcos[1] = 2.0f*cos_angle*normal[1] + dcos[1];
				dcos[2] = 2.0f*cos_angle*normal[2] + dcos[2];
		}
		else if (flag_ref == 1)				// transmission	
		{
			 dcos[0]= -normal[0]*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos[0]+(cos_angle*normal[0]));
			 dcos[1]= -normal[1]*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos[1]+(cos_angle*normal[1]));
			 dcos[2]= -normal[2]*cos(trans_theta)-(sin(trans_theta)/sin(refl_theta))*(dcos[2]+(cos_angle*normal[2]));
		}

		// normalize
		norm = sqrt(dcos[0]*dcos[0] + dcos[1]*dcos[1] + dcos[2]*dcos[2]);

		if ((norm < (1.0f - epsilon)) || (norm > (1.0f + epsilon)))
		 {
			dcos[0] = dcos[0]/norm;
			dcos[1] = dcos[1]/norm;
			dcos[2] = dcos[2]/norm;
		 } 

	return;	
	}

#endif

/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// add roughness to the surface of the column according to roughness coefficient 'beta'
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA

	__device__ inline void RoughSurface(float3 *normal, int2* seed, float beta)
	{

		float theta = 0.0f;
		float status = 0.0f;
		float rr = 0.0f;
		float3 normalpert = {0.0f};
		float3 rough_normal = {0.0f};
		float normalize_base = 0.0f;

		// generate the perturbation vector
		status = ranecu(seed);
		normalpert.z = 2.0f*status - 1.0f;	// random number between (-1,1)
		rr = sqrt(1.0f - status*status);
		status = ranecu(seed);
		theta = status * 2.0f * pi;		// random number between (0,2pi)

		normalpert.x = rr * cos(theta);
		normalpert.y = rr * sin(theta);

		// normalize the perturbed vector
		normalize_base = sqrt( pow(normalpert.x,2) + pow(normalpert.y,2) + pow(normalpert.z,2) );
	
		normalpert.x = normalpert.x/normalize_base;
		normalpert.y = normalpert.y/normalize_base;
		normalpert.z = normalpert.z/normalize_base;

		// rough normal = beta*perturbed + original normal
		rough_normal.x = beta * normalpert.x + normal->x;	
		rough_normal.y = beta * normalpert.y + normal->y;
		rough_normal.z = beta * normalpert.z + normal->z;

		// normalize rough normal
		normalize_base = sqrt( pow(rough_normal.x,2) + pow(rough_normal.y,2) + pow(rough_normal.z,2) );

		normal->x = rough_normal.x/normalize_base; 
		normal->y = rough_normal.y/normalize_base;
		normal->z = rough_normal.z/normalize_base;

	return;
	}
	
#else	// C code

	void RoughSurface(float *normal, int* seed, float beta)
	{
		float theta = 0.0f;
		float status = 0.0f;
		float rr = 0.0f;
		float normalpert[3] = {0.0f};
		float rough_normal[3] = {0.0f};
		float normalize_base = 0.0f;

		// generate the perturbation vector
		status = ranecu(seed);
		normalpert[2] = 2.0f*status - 1.0f;	// random number between (-1,1)
		rr = sqrt(1.0f - status*status);
		status = ranecu(seed);
		theta = status * 2.0f * pi;		// random number between (0,2pi)

		normalpert[0] = rr * cos(theta);
		normalpert[1] = rr * sin(theta);

		// normalize the perturbed vector
		normalize_base = sqrt( pow(normalpert[0],2) + pow(normalpert[1],2) + pow(normalpert[2],2) );
	
		normalpert[0] = normalpert[0]/normalize_base;
		normalpert[1] = normalpert[1]/normalize_base;
		normalpert[2] = normalpert[2]/normalize_base;

		// rough normal = beta*perturbed + original normal
		rough_normal[0] = beta * normalpert[0] + normal[0];	
		rough_normal[1] = beta * normalpert[1] + normal[1];
		rough_normal[2] = beta * normalpert[2] + normal[2];

		// normalize rough normal
		normalize_base = sqrt( pow(rough_normal[0],2) + pow(rough_normal[1],2) + pow(rough_normal[2],2) );

		normal[0] = rough_normal[0]/normalize_base; 
		normal[1] = rough_normal[1]/normalize_base;
		normal[2] = rough_normal[2]/normalize_base;

	return;
	}	

#endif


/////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// determine if the photon gets detected
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
__device__ inline int detection(float3 *pos, float H, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, 
unsigned long long int *myimage, size_t pitch, struct start_info *info, int mytid, int *num_detected_primary, float sensorRefl, float d_min, int2* seed, 
float3 *dcos, float3 *normal, float bulk_abscoeff, float R, float xdetector, float ydetector, unsigned long long int mynum_rebound, float *Xc, float *Yc,
struct histStruct * dPhotonHist, int boolToCollectData)
{
	int result = 0;		// flag - photon detected at sensor plane or absorbed in the bulk
	int absflag = 0;	// flag - photon absorbed in the bulk (yes=1; no=0); returned from isotropic
	int ii = 0, jj = 0;	// x,y pixel number for detected photons
		
	/*if(mytid == 6 || mytid == 7)
	{
			printf("tid: %d in detection\n");
	}*/
		
	// if z = -H/2 (sensor plane), photon detected
	if (fabs(pos->z - (float)(-H/2.0f)) < epsilon) 
	{
		if(ranecu(seed) < sensorRefl)	// non-ideal sensor - reflects back sensorRefl% of photons into the current column; detects rest
		{
			// normal pointing +z-direction
			normal->x = 0.0f; 
			normal->y = 0.0f; 
			normal->z = 1.0f;

			// reflected dir. cosine from the sensor plane (bottom surface) (bottom surface is smooth, no need to perturb the normal; specular reflection)
			trans_dir_cos(dcos, normal, 0.0f, 0.0f, 0, mytid, info); // reflection only, so 'refl_theta,trans_theta' = 0

			// using above calculated dir. cosine, move the photon within the column
			absflag = isotropic(pos, dcos, seed, bulk_abscoeff, R, H, xdetector, ydetector, &info[mytid], mynum_rebound, Xc, Yc, mytid, dPhotonHist, boolToCollectData);

			if(absflag == 1)	// absorbed in bulk
				result = 1;
			else
				result = 0;
		}
		else				// photon detected  
		{
			result = 1;
			atomicAdd(&num_detect, 1);	// increment photon detected counter

			if(mytid < dev_numPhotonHist && boolToCollectData == 1)
			{
				dPhotonHist[mytid].x[dPhotonHist[mytid].histCounter] = pos->x;
				dPhotonHist[mytid].y[dPhotonHist[mytid].histCounter] = pos->y;
				dPhotonHist[mytid].z[dPhotonHist[mytid].histCounter] = pos->z;
				dPhotonHist[mytid].Xc[dPhotonHist[mytid].histCounter] = *Xc;
				dPhotonHist[mytid].Yc[dPhotonHist[mytid].histCounter] = *Yc;
				dPhotonHist[mytid].terminated[dPhotonHist[mytid].histCounter] = 1;
				dPhotonHist[mytid].histCounter ++;
			}
			
			/*if(mytid == 6 || mytid == 7)
			{
				printf("detection - tid: %d, %f %f %f %f %f\n", mytid, pos->x, pos->y, pos->z, *Xc, *Yc);
			}*/
			
			ii = floor((pos->x-lbound_x)/pixelsize);	// x,y pixel number in the sensor plane of the detector
			jj = floor((pos->y-lbound_y)/pixelsize);

			// if the photon gets detected within lower and upper bounds of PRF: accumulate the signal contribution
			if( (pos->x <= ubound_x) && (pos->y <= ubound_y) && (pos->x >= lbound_x) && (pos->y >= lbound_y) )
			{	
				unsigned long long int* current_img = (unsigned long long int*)((char*)myimage + ii * pitch);
				atomicAdd(&current_img[jj],1);
			}
			atomicAdd(&num_detected_primary[info[mytid].str_histnum-1],1);	// increment # detected per primary
		} 
	}
	else
		result = 0;

	return result;
}	
#else	// C code

	int detection(float *pos, float H, int pixelsize, float lbound_x, float lbound_y, float ubound_x, float ubound_y, unsigned long long int *myimage, struct start_info info, 
	float sensorRefl, float d_min, int* seed, float *dcos, float *normal, float bulk_abscoeff, float R, float xdetector, float ydetector, unsigned long long int mynum_rebound, 
	int ydim, int *h_num_detected_prim)
	{

		int result = 0;		// flag - photon detected at sensor plane or absorbed in the bulk
		int absflag = 0;	// flag - photon absorbed in the bulk (yes=1; no=0); returned from isotropic
		int ii = 0, jj = 0;	// x,y pixel number for detected photons


		// if z = -H/2 (sensor plane), photon detected
		if (fabs(pos[2] - (float)(-H/2.0f)) < epsilon) 
		 {
			if(ranecu(seed) < sensorRefl)		// non-ideal sensor - reflects back sensorRefl% of photons into the current column; detects rest
			{

				// normal pointing +z-direction
				normal[0] = 0.0f; 
				normal[1] = 0.0f; 
				normal[2] = 1.0f;

				// reflected dir. cosine from the sensor plane (bottom surface) (bottom surface is smooth, no need to perturb the normal; specular reflection)
				trans_dir_cos(dcos, normal, 0.0f, 0.0f, 0, info); // reflection only, so 'refl_theta, trans_theta' = 0

				// using above calculated dir. cosine, move the photon within the column
				absflag = isotropic(pos, dcos, seed, bulk_abscoeff, R, H, xdetector, ydetector, info, mynum_rebound);

				if(absflag == 1)	// absorbed in bulk
					result = 1;
				else
					result = 0;

			}
			else					// photon detected  
			{
				result = 1;
				num_detect++;
		
				ii = floor((pos[0]-lbound_x)/pixelsize);	// x,y pixel number in the sensor plane of the detector
				jj = floor((pos[1]-lbound_y)/pixelsize);

				// if the photon gets detected within lower and upper bounds of PRF: accumulate the signal contribution
				if( (pos[0] <= ubound_x) && (pos[1] <= ubound_y) && (pos[0] >= lbound_x) && (pos[1] >= lbound_y) )
				 {	
					outputimage_.newimageopt[ii][jj]++;
				 }

				h_num_detected_prim[info.str_histnum]++;	// increment # detected per primary
			}
			 
		 }
		else
		    	result = 0;
	  
	 return result;
	}

#endif


////////////////////////////////////////////////////////////////////////////////
//! Initialize the pseudo-random number generator (PRNG) RANECU to a position
//! far away from the previous history (leap frog technique).
//!
//! Each calculated seed initiates a consecutive and disjoint sequence of
//! pseudo-random numbers with length LEAP_DISTANCE, that can be used to
//! in a parallel simulation (Sequence Splitting parallelization method).
//! The basic equation behind the algorithm is:
//!    S(i+j) = (a**j * S(i)) MOD m = [(a**j MOD m)*S(i)] MOD m  ,
//! which is described in:
//!   P L'Ecuyer, Commun. ACM 31 (1988) p.742
//!
//! This function has been adapted from "seedsMLCG.f", see:
//!   A Badal and J Sempau, Computer Physics Communications 175 (2006) p. 440-450
//!
//!       @param[in] history   Particle bach number.
//!       @param[in] seed_input   Initial PRNG seed input (used to initiate both MLCGs in RANECU).
//!       @param[out] seed   Initial PRNG seeds for the present history.
//!
////////////////////////////////////////////////////////////////////////////////
// -- Upper limit of the number of random values sampled in a single track:
#define  LEAP_DISTANCE    1000
// -- Multipliers and moduli for the two MLCG in RANECU:
#define  a1_RANECU       40014
#define  m1_RANECU  2147483563
#define  a2_RANECU       40692
#define  m2_RANECU  2147483399

#ifdef USING_CUDA
	__device__ inline void init_PRNG(int history_batch, int histories_per_thread, int seed_input, int2* seed)
	{
	  // -- Move the RANECU generator to a unique position for the current batch of histories:
	  //    I have to use an "unsigned long long int" value to represent all the simulated histories in all previous batches
	  //    The maximum unsigned long long int value is ~1.8e19: if history >1.8e16 and LEAP_DISTANCE==1000, 'leap' will overflow.
	  // **** 1st MLCG:
	  unsigned long long int leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
	  int y = 1;
	  int z = a1_RANECU;
	  // -- Calculate the modulo power '(a^leap)MOD(m)' using a divide-and-conquer algorithm adapted to modulo arithmetic
	  for(;;)
	  {
	    // (A2) Halve n, and store the integer part and the residue
	    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  Equivalent: t=(short)(leap%2);
	    {
	      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2);  
	      y = abMODm(m1_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
	      if (0==leap) break;         // (A4) leap==0? ==> finish
	    }
	    else           // (leap is even)
	    {
	      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	    }
	    z = abMODm(m1_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
	  }
	  // AjMODm1 = y;                 // Exponentiation finished:  AjMODm = expMOD = y = a^j

	  // -- Compute and display the seeds S(i+j), from the present seed S(i), using the previously calculated value of (a^j)MOD(m):
	  //         S(i+j) = [(a**j MOD m)*S(i)] MOD m
	  //         S_i = abMODm(m,S_i,AjMODm)
	  seed->x = abMODm(m1_RANECU, seed_input, y);     // Using the input seed as the starting seed

	  // **** 2nd MLCG (repeating the previous calculation for the 2nd MLCG parameters):
	  leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
	  y = 1;
	  z = a2_RANECU;
	  for(;;)
	  {
	    // (A2) Halve n, and store the integer part and the residue
	    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  Equivalent: t=(short)(leap%2);
	    {
	      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	      y = abMODm(m2_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
	      if (0==leap) break;         // (A4) leap==0? ==> finish
	    }
	    else           // (leap is even)
	    {
	      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	    }
	    z = abMODm(m2_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
	  }
	  // AjMODm2 = y;
	  seed->y = abMODm(m2_RANECU, seed_input, y);     // Using the input seed as the starting seed
	}
#else
	void init_PRNG(int history_batch, int histories_per_thread, int seed_input, int* seed)
	{
	  // -- Move the RANECU generator to a unique position for the current batch of histories:
	  //    I have to use an "unsigned long long int" value to represent all the simulated histories in all previous batches
	  //    The maximum unsigned long long int value is ~1.8e19: if history >1.8e16 and LEAP_DISTANCE==1000, 'leap' will overflow.
	  // **** 1st MLCG:
	  unsigned long long int leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
	  int y = 1;
	  int z = a1_RANECU;
	  // -- Calculate the modulo power '(a^leap)MOD(m)' using a divide-and-conquer algorithm adapted to modulo arithmetic
	  for(;;)
	  {
	      // printf(" leap, leap>>1, leap&1: %d, %d, %d\n",leap, leap>>1, leap&1);  

	    // (A2) Halve n, and store the integer part and the residue
	    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  !!DeBuG!! OLD: t=(short)(leap%2);
	    {
	      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2); 
	      y = abMODm(m1_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
	      if (0==leap) break;         // (A4) leap==0? ==> finish
	    }
	    else           // (leap is even)
	    {
	      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	    }
	    z = abMODm(m1_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
	  }
	  // AjMODm1 = y;                 // Exponentiation finished:  AjMODm = expMOD = y = a^j

	  // -- Compute and display the seeds S(i+j), from the present seed S(i), using the previously calculated value of (a^j)MOD(m):
	  //         S(i+j) = [(a**j MOD m)*S(i)] MOD m
	  //         S_i = abMODm(m,S_i,AjMODm)
	  seed[0] = abMODm(m1_RANECU, seed_input, y);     // Using the input seed as the starting seed

	  // **** 2nd MLCG (repeating the previous calculation for the 2nd MLCG parameters):
	  leap = ((unsigned long long int)(history_batch+1))*(histories_per_thread*LEAP_DISTANCE);
	  y = 1;
	  z = a2_RANECU;
	  for(;;)
	  {
	    // (A2) Halve n, and store the integer part and the residue
	    if (0!=(leap&01))  // (bit-wise operation for MOD(leap,2), or leap%2 ==> proceed if leap is an odd number)  !!DeBuG!! OLD: t=(short)(leap%2);
	    {
	      leap >>= 1;     // Halve n moving the bits 1 position right. Equivalent to:  leap=(leap/2); 
	      y = abMODm(m2_RANECU,z,y);      // (A3) Multiply y by z:  y = [z*y] MOD m
	      if (0==leap) break;         // (A4) leap==0? ==> finish
	    }
	    else           // (leap is even)
	    {
	      leap>>= 1;     // Halve leap moving the bits 1 position right. Equivalent to:  leap=(leap/2);
	    }
	    z = abMODm(m2_RANECU,z,z);        // (A5) Square z:  z = [z*z] MOD m
	  }
	  // AjMODm2 = y;
	  seed[1] = abMODm(m2_RANECU, seed_input, y);     // Using the input seed as the starting seed

	}
#endif


/////////////////////////////////////////////////////////////////////
//!  Calculate "(a1*a2) MOD m" with 32-bit integers and avoiding   **
//!  the possible overflow, using the Russian Peasant approach     **
//!  modulo m and the approximate factoring method, as described   **
//!  in:  L'Ecuyer and Cote, ACM Trans. Math. Soft. 17 (1991)      **
//!                                                                **
//!  This function has been adapted from "seedsMLCG.f", see:       **
//!  Badal and Sempau, Computer Physics Communications 175 (2006)  **
//!                                                                **
//!    Input:          0 < a1 < m                                  **
//!                    0 < a2 < m                                  **
//!                                                                **
//!    Return value:  (a1*a2) MOD m                                **
//!                                                                **
/////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline int abMODm(int m, int a, int s)
	{
	  // CAUTION: the input parameters are modified in the function but should not be returned to the calling function! (pass by value!)
	  int q, k;
	  int p = -m;            // p is always negative to avoid overflow when adding

	  // ** Apply the Russian peasant method until "a =< 32768":
	  while (a>32768)        // We assume '32' bit integers (4 bytes): 2^(('32'-2)/2) = 32768
	  {
	    if (0!=(a&1))        // Store 's' when 'a' is odd     Equivalent code:   if (1==(a%2))
	    {
	      p += s;
	      if (p>0) p -= m;
	    }
	    a >>= 1;             // Half a (move bits 1 position right)   Equivalent code: a = a/2;
	    s = (s-m) + s;       // Double s (MOD m)
	    if (s<0) s += m;     // (s is always positive)
	  }

	  // ** Employ the approximate factoring method (a is small enough to avoid overflow):
	  q = (int) m / a;
	  k = (int) s / q;
	  s = a*(s-k*q)-k*(m-q*a);
	  while (s<0)
	    s += m;

	  // ** Compute the final result:
	  p += s;
	  if (p<0) p += m;

	  return p;
	}
#else
	int abMODm(int m_par, int a_par, int s_par)
	{
	  // CAUTION: the input parameters are modified in the function but should not be returned to the calling function! (pass by value!)   !!DeBuG!!
	  int mval,aval,sval;
	  mval=m_par; aval=a_par; sval=s_par;
	  
	  int qval, kval;
	  int pval = -mval;            // p is always negative to avoid overflow when adding

	  // ** Apply the Russian peasant method until "a =< 32768":
	  while (aval>32768)        // We assume '32' bit integers (4 bytes): 2^(('32'-2)/2) = 32768
	  {
	    if (0!=(aval&1))        // Store 's' when 'a' is odd    !!DeBuG!! OLD code:   if (1==(a%2))
	    {
	      pval += sval;
	      if (pval>0) pval -= mval;
	    }
	    aval >>= 1;             // Half a (move bits 1 position right)        
	    sval = (sval-mval) + sval;       // float s (MOD m)
	    if (sval<0) sval += mval;     // (s is always positive)
	  }

	  // ** Employ the approximate factoring method (a is small enough to avoid overflow):
	  qval = (int) mval / aval;
	  kval = (int) sval / qval;
	  sval = aval*(sval-kval*qval)-kval*(mval-qval*aval);
	  while (sval<0)
	    sval += mval;

	  // ** Compute the final result:
	  pval += sval;
	  if (pval<0) pval += mval;

	  return pval;
	}
#endif


////////////////////////////////////////////////////////////////////////////////
//! Pseudo-random number generator (PRNG) RANECU returning a float value
//! (single precision version).
//!
//!       @param[in,out] seed   PRNG seed (seed kept in the calling function and updated here).
//!       @return   PRN double value in the open interval (0,1)
//!
////////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA
	__device__ inline float ranecu(int2* seed)
	{
	//return (float(seed->x%100)*0.01f+0.005f)  ;

	  int i1 = (int)(seed->x/53668);
	  seed->x = 40014*(seed->x-i1*53668)-i1*12211;

	  int i2 = (int)(seed->y/52774);
	  seed->y = 40692*(seed->y-i2*52774)-i2*3791;

	  if (seed->x < 0) seed->x += 2147483563;
	  if (seed->y < 0) seed->y += 2147483399;

	  i2 = seed->x-seed->y;
	  if (i2 < 1) i2 += 2147483562;

	  return (__int2float_rn(i2)*4.65661305739e-10f);        // 4.65661305739e-10 == 1/2147483563

	}
#else
	float ranecu(int* seed)
	{
	  int i1 = (int)(seed[0]/53668);
	  seed[0] = 40014*(seed[0]-i1*53668)-i1*12211;

	  int i2 = (int)(seed[1]/52774);
	  seed[1] = 40692*(seed[1]-i2*52774)-i2*3791;

	  if (seed[0] < 0) seed[0] += 2147483563;
	  if (seed[1] < 0) seed[1] += 2147483399;

	  i2 = seed[0]-seed[1];
	  if (i2 < 1) i2 += 2147483562;

	  const float USCALE = 1.0/2147483563.0;       
	  return ((float)(i2*USCALE));

	}
#endif

