#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
//
// 			     //////////////////////////////////////////////////////////
//  			     //							     //
// 			     //   	        hybridMANTIS v1.0		     //
//			     //	    	    fastDETECT2  - CUDA code 		     //
//			     //		   (optical photons transport)		     //
//			     //							     //
//			     //////////////////////////////////////////////////////////
//
// 
//
// 
//
//
// ****Disclaimer****
//  This software and documentation (the "Software") were developed at the Food and Drug Administration (FDA) by employees of the Federal Government in
//  the course of their official duties. Pursuant to Title 17, Section 105 of the United States Code, this work is not subject to copyright protection
//  and is in the public domain. Permission is hereby granted, free of charge, to any person obtaining a copy of the Software, to deal in the Software
//  without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, or sell copies of the
//  Software or derivatives, and to permit persons to whom the Software is furnished to do so. FDA assumes no responsibility whatsoever for use by other
//  parties of the Software, its source code, documentation or compiled executables, and makes no guarantees, expressed or implied, about its quality,
//  reliability, or any other characteristic. Further, use of this code in no way implies endorsement by the FDA or confers any advantage in regulatory
//  decisions. Although this software can be redistributed and/or modified freely, we ask that any derivative works bear some notice that they are
//  derived from it, and any modified versions bear some notice that they have been modified. 
//
//
//	Associated publication: Sharma Diksha, Badal Andreu and Badano Aldo, "hybridMANTIS: a CPU-GPU Monte Carlo method for modeling indirect x-ray detectors with
//				columnar scintillators". Physics in Medicine and Biology, 57(8), pp. 2357–2372 (2012)
//
//
//	File:   	hybridMANTIS_cuda_ver1_0.cu 			
//	Author: 	Diksha Sharma (US Food and Drug Administration)
//	Email: 		diksha.sharma@fda.hhs.gov			
//	Last updated:  	Apr 18, 2012
// 
//	Modified Name:	visualMANTIS_cuda_ver1_0.cu
//	Updated: 	4/23/2013
//	Author:		Han Dong (US Food and Drug Administration / University of Maryland Baltimore County
//	Email:		han6@umbc.edu
//	Comments:	This code contains modified C code that called CUDA kernels in order to retrieve photon information during execution of hybridMANTIS.
//			The algorithm and code is unchanged and thus is the same as the base hybridMANTIS code, however what was added was
//			additional data structures to save photon data during the execution so that it can be used by the visualization for
//			rendering. At the end of each iteration, the data structures are read back from the GPU and parsed and saved to text
//			files that are then used for visualization.
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////
//
//      Header libraries
//
/////////////////////////////////////////
#include <gsl/gsl_rng.h>
#include <gsl/gsl_randist.h>

#ifdef USING_CUDA
	#include <cutil_inline.h>
	#include <hip/hip_vector_types.h>
	#include <stdint.h>
#endif

#include <unistd.h>
#include <stdarg.h>

/////////////////////////////////////////
//
//      Global variables
//
/////////////////////////////////////////
#define max_photon_per_EDE 900000	// maximum number of optical photons that can be generated per energy deposition event (EDE)

#ifdef USING_CUDA
	#define gpubufsize 2304000	// GPU buffer size: # of events sent to the GPU
#endif

/////////////////////////////////////////
//
//      Include kernel program
//
/////////////////////////////////////////
#include "visual_kernel_cuda_c_ver1_0.cu"

/////////////////////////////////////////
//
//      CUDA parameters
//
/////////////////////////////////////////
#ifdef USING_CUDA
	#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
	printf("Error at %s:%d\n",__FILE__,__LINE__); \
	return EXIT_FAILURE;}} while(0)

	#define GRIDSIZE 18000		// number of blocks
	#define BLOCKSIZE 128		// number of threads
#endif

int counter = 0;
unsigned long long int *han_h_myimage;
unsigned long long int *han_h_photonHist;
unsigned long long int *han_d_photonHist;
int * vis_num_detected_primary;
int numPhotonHist;
int finishPhotonHist = 0;
int boolToCollectData = 0;

////////////////////////////////////////////////////////////////////////////
//				MAIN PROGRAM			          //
////////////////////////////////////////////////////////////////////////////

#ifdef USING_CUDA

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// gpuoptical(): Performs optical transport using GPU 
//	  	 Input arguments: penctr, myfactGPU
//
// 		 penctr - flag to indicate how optical transport will be run on GPU. 
//		   a value of '99' : calling gpuoptical() first time to initialize the GPU and allocate memories and reset counters.
//		   a value of '100': calling optical transport kernel
//		   a value of '101': calling gpuoptical() last time; running optical transport for remaining buffer; copying data from device to host and getting output images.
// 		 myfactGPU - buffer size to be sent to GPU after load balancing
//
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	extern "C" void gpuoptical_(int *penctr, int *myfactGPU)	
	{    
		// command line arguments
		float xdetector, ydetector, radius, height, n_C, n_IC, top_absfrac, bulk_abscoeff, beta, d_min, lbound_x, lbound_y, ubound_x, ubound_y, d_max, yield, sensorRefl;
		int pixelsize, num_primary, min_optphotons, max_optphotons, num_bins;
		int i, j;
		
		// CUDA variables	
		unsigned int hTimer = 0;		// timer
		static float totalgpuTime = 0.0f;	// total time taken
		dim3 threads, blocks;			// threads and blocks
		int devID;				// GPU device ID
		
		// GNU scientific library (gsl) variables
		const gsl_rng_type * Tgsl;
		gsl_rng * rgsl;
		double mu_gsl;	
		
		// Host (CPU) counters		
		unsigned long long int host_num_generated = 0; 	// total # of photons generated for all the x-ray histories (across all threads)
		unsigned long long int host_num_detect = 0;	// total # of photons detected at the sensor plane of a column
		unsigned long long int host_num_lost = 0;	// total # of photons lost when exiting out of the detector boundaries in x/y direction
		unsigned long long int host_num_abs_top = 0;	// total # of photons absorbed at the top surface of the detector
		unsigned long long int host_num_abs_bulk = 0;	// total # of photons absorbed in the bulk of the detector
		unsigned long long int host_num_outofcol = 0;	// total # of photons killed because they moved out of current column when reflected (due to precision errors)
		unsigned long long int host_num_theta1 = 0;	// total # of photons killed if incidence angle > 1.57 or < 0 radian (after resampling max 100 times)	

		// create 2D array for storing output PRF image
		int xdim = 0;
		int ydim = 0;
		int indexi=0, indexj=0;
		int my_index=0;
		size_t pitch;				// pitch used for storing 2D image array in GPU memory

		int nbytes = (*myfactGPU)*sizeof(struct start_info);	// total number of bytes for storing interaction events buffer information

		// allocate memory pointers
		unsigned long long int *myimage = 0;	// device memory for output image
		int *num_detected_primary = 0;		// device memory for # detected photons/primary
		struct start_info *h_a = 0;             // pointer to the struct info data in the host memory
		struct start_info *d_a = 0;             // pointers to struct data in the device memory

		// copy to local variables from PENELOPE common block
		xdetector = inputargs_.detx;		// x dimension of detector (in um). x in (0,xdetector)
		ydetector = inputargs_.dety;		// y dimension of detector (in um). y in (0,ydetector)
		height = inputargs_.detheight;		// height of column and thickness of detector (in um). z in range (-H/2, H/2)
		radius = inputargs_.detradius;		// radius of column (in um).
		n_C = inputargs_.detnC;			// refractive index of columns
		n_IC = inputargs_.detnIC;		// refractive index of intercolumnar material
		top_absfrac = inputargs_.dettop;	// column's top surface absorption fraction (0,1)
		bulk_abscoeff = inputargs_.detbulk;	// column's bulk absorption coefficient (in um^-1) 
		beta = inputargs_.detbeta;		// roughness coefficient of column walls (0,0.5)
		d_min = inputargs_.detdmin;		// minimum distance a photon can travel when transmitted from a column
		d_max = inputargs_.detdmax;		// maximum distance a photon can travel when transmitted from a column
		lbound_x = inputargs_.detlboundx;	// x lower bound of region of interest of output PRF image (in um)
		lbound_y = inputargs_.detlboundy;	// y lower bound (in um)
		ubound_x = inputargs_.detuboundx;	// x upper bound (in um) 
		ubound_y = inputargs_.detuboundy;	// y upper bound (in um)
		yield = inputargs_.detyield;		// light yield (/eV)
		pixelsize = inputargs_.detpixel;	// 1 pixel = 'pixelsize' microns (in um)
		sensorRefl = inputargs_.detsensorRefl;	// Non-Ideal sensor reflectivity (%) (0,100)
		num_primary = inputargs_.mynumhist;	// total number of primaries to be simulated
		min_optphotons = inputargs_.minphotons;	// minimum number of optical photons detected to be included in PHS
		max_optphotons = inputargs_.maxphotons;	// maximum number of optical photons detected to be included in PHS
		num_bins = inputargs_.mynumbins;	// number of bins for genrating PHS
				
		// set the device with max GFlops	
		devID = cutGetMaxGflopsDeviceId();
		hipSetDevice( devID );

	    // create a generator chosen by the 
		//  environment variable GSL_RNG_TYPE 
	    gsl_rng_env_setup();	     
	    Tgsl = gsl_rng_default;
	    rgsl = gsl_rng_alloc (Tgsl);

		// dimensions of PRF image
		xdim = ceil((ubound_x - lbound_x)/pixelsize);
		ydim = ceil((ubound_y - lbound_y)/pixelsize);

		//han
		unsigned long long int *h_myimage = 0;          // page-locked host memory for asynchronous copying (contain host image for evey kernel run)
		int *h2_num_detected_primary = 0;
		struct histStruct * hPhotonHist;
		struct histStruct * dPhotonHist;
		printf("%d\n", *penctr);

		if(*penctr == 99)	// initialize GPU; allocate and initialize memories
		{
			// allocate device memory for storing output arrays
			hipMallocPitch((void**)&myimage, &pitch, xdim*sizeof(unsigned long long int), ydim);		// allocate 2D image array (PRF)
			cutilSafeCall( hipMemset2D(myimage, pitch, 0, xdim*sizeof(unsigned long long int), ydim) );	// initialize to 0

			cutilSafeCall( hipMalloc((void**)&num_detected_primary, sizeof(int)*num_primary) );		// create 1D array for outputting # detected/primary
			cutilSafeCall( hipMemset(num_detected_primary, 0, sizeof(int)*num_primary) );			// initialize to 0

			// allocate host and device memory for transferring buffer information
			cutilSafeCall( hipHostMalloc((void**)&h_a, nbytes) ); 
			cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );

			// copy address of memory pointers to PENELOPE variables. These variables are used later to point to device and host memories without re-initializing the GPU.
			gpumemaddr_.gpuimage 	= (unsigned long long int)myimage;
			gpumemaddr_.gpudetect 	= (unsigned long long int)num_detected_primary;
			gpumemaddr_.hosta 	= (unsigned long long int)h_a;	
			gpumemaddr_.deva 	= (unsigned long long int)d_a;
		    gpumemaddr_.devpitch 	= (unsigned long long int)pitch;

			// reset the host counters
			host_num_generated=0;
			host_num_detect=0;
			host_num_abs_top=0;	
			host_num_abs_bulk=0;	
			host_num_lost=0;
			host_num_outofcol=0;
			host_num_theta1=0;

			FILE * fphist;
			char numPhotonHistStr[100];
			fphist = fopen("numPhotonHistories.txt", "r");
			fgets(numPhotonHistStr, 100, fphist);
			numPhotonHist = atoi(numPhotonHistStr);
			fclose(fphist);
			
			// reset device counters to zero
			cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_detect"),&host_num_detect,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));	
			cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_generated"),&host_num_generated,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));	
			cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_abs_top"),&host_num_abs_top,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));	
			cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_abs_bulk"),&host_num_abs_bulk,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));	
			cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_lost"),&host_num_lost,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));
			cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_outofcol"),&host_num_outofcol,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));
			cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("num_theta1"),&host_num_theta1,sizeof(unsigned long long int)*1,0,hipMemcpyHostToDevice));
			cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("dev_numPhotonHist"), &numPhotonHist, sizeof(int)*1, 0 , hipMemcpyHostToDevice));

			//han
			//allocate memory for h_myimage
			cutilSafeCall( hipHostMalloc((void**)&h_myimage, xdim*ydim*sizeof(unsigned long long int)) ); 
			//allocate memory for h2_num_detected_primary
			cutilSafeCall( hipHostMalloc((void**)&h2_num_detected_primary, sizeof(int) * num_primary) ); 
			
			// allocate host and device memory for transferring buffer information in photon histories
			cutilSafeCall( hipHostMalloc((void**)&hPhotonHist, numPhotonHist*sizeof(struct histStruct)) ); 
			cutilSafeCall( hipMalloc((void**)&dPhotonHist, numPhotonHist*sizeof(struct histStruct)) );
			
			for(i=0;i<numPhotonHist;i++)
			{
				hPhotonHist[i].histCounter = 0;
			}
			cutilSafeCall( hipMemcpy(dPhotonHist, hPhotonHist, numPhotonHist*sizeof(struct histStruct), hipMemcpyHostToDevice) );
			
			// copy memory pointers to avoid re mallocing
			han_h_myimage = h_myimage;
			han_h_photonHist = (unsigned long long int *)hPhotonHist;
			han_d_photonHist = (unsigned long long int *)dPhotonHist;
			vis_num_detected_primary = h2_num_detected_primary;
			
			cutilCheckError( cutCreateTimer(&hTimer) );

		}
		else if(*penctr == 100)			// run optical kernel
		{
			FILE *fp;
			char detectName[100];
			char filename[100];
			char pHistName[100];
				
			// synchronize threads to ensure that GPU is not busy processing previous kernel call
			hipDeviceSynchronize();
			hPhotonHist = (struct histStruct *) han_h_photonHist;
			dPhotonHist = (struct histStruct *) han_d_photonHist;
			
			if(counter == 0)
			{
				counter +=1;
			}
			else
			{
				h_myimage = han_h_myimage;
				h2_num_detected_primary = vis_num_detected_primary;
				
				sprintf(filename, "myimage%d.dat", counter);
				fp=fopen(filename, "w");
				
				// add h_myimage to the new_myimage (array in PENELOPE)
				for(indexi = 0; indexi < ydim; indexi++)
				{
		 	 		for(indexj = 0; indexj < xdim; indexj++)
					{
						outputimage_.newimageopt[indexi][indexj] = outputimage_.newimageopt[indexi][indexj] + h_myimage[indexi*xdim + indexj];
						fprintf(fp, "       %.4f\n",  outputimage_.newimageopt[indexi][indexj] * (1.0 / optical_.nbufopt[(*myfactGPU)-1]));
					}
					fprintf(fp, "\n");
				}
				fclose(fp);
				
				/*for(i=0;i<numPhotonHist;i++)
				{
					sprintf(pHistName, "photon_hist_%d_%d.dat", counter, i);
				
					fp = fopen(pHistName, "w");	
					fprintf(fp, "%d\n", hPhotonHist[i].histCounter);
					for(j=0;j<hPhotonHist[i].histCounter;j++)
					{
						fprintf(fp, "%f %f %f %f %f %d\n", hPhotonHist[i].x[j], hPhotonHist[i].y[j], hPhotonHist[i].z[j], hPhotonHist[i].Xc[j], hPhotonHist[i].Yc[j], hPhotonHist[i].terminated[j]);
					}
					
					fclose(fp);
				}*/
				
				int indexi, indexj;
				int *h_histogram = 0;		// host memory for storing histogram of # photons detected/primary
				h_histogram = (int*)malloc(sizeof(int)*num_bins);
				
				for(indexj=0; indexj < num_bins; indexj++)
				{
					h_histogram[indexj] = 0;
				}
				
				// make histogram of number of detected photons/primary for num_bins
				int binsize=0, newbin=0;
				int bincorr=0;
							
				binsize = floor((max_optphotons-min_optphotons)/num_bins);	// calculate size of each bin. Assuming equally spaced bins.
				bincorr = floor(min_optphotons/binsize);			// correction in bin number if min_optphotons > 0.
			
				for(indexi = 0; indexi < num_primary; indexi++)
				{
					newbin = floor(h2_num_detected_primary[indexi]/binsize) - bincorr;	// find bin #
			 	
					if(h2_num_detected_primary[indexi] > 0)	// store only non-zero bins
					{
						if(h2_num_detected_primary[indexi] <= min_optphotons)	// # detected < minimum photons given by user, add to the first bin
							h_histogram[0]++;
						else if(h2_num_detected_primary[indexi] >= max_optphotons)	// # detected > maximum photons given by user, then add to the last bin
							h_histogram[num_bins-1]++;
						else
							h_histogram[newbin]++; 
					}
				}
				
				sprintf(detectName, "detect%d.dat", counter);
				fp=fopen(detectName, "w");
				
				// add num_detected_primary to gldetprimary array in PENELOPE
				for(indexi = 0; indexi < num_bins; indexi++)
				{
					outputdetprim_.gldetprimary[indexi] = outputdetprim_.gldetprimary[indexi] + h_histogram[indexi];
					fprintf(fp, "        %d\n", outputdetprim_.gldetprimary[indexi]);
				}
				fclose(fp);
				
				counter += 1;
				printf("Finished writing photon history and image data.\n");
			}
			
			if(finishPhotonHist < numPhotonHist)
			{
				for(i=0;i<numPhotonHist;i++)
				{
					if(hPhotonHist[i].histCounter > 0 && finishPhotonHist < numPhotonHist)
					{
						sprintf(pHistName, "photon_hist_1_%d.dat", finishPhotonHist);
				
						fp = fopen(pHistName, "w");	
						fprintf(fp, "%d\n", hPhotonHist[i].histCounter);
						for(j=0;j<hPhotonHist[i].histCounter;j++)
						{
							fprintf(fp, "%f %f %f %f %f %d\n", hPhotonHist[i].x[j], hPhotonHist[i].y[j], hPhotonHist[i].z[j], hPhotonHist[i].Xc[j], hPhotonHist[i].Yc[j], hPhotonHist[i].terminated[j]);
						}
					
						finishPhotonHist += 1;
						fclose(fp);
					}
				}
			}
			
			// allocate nbytes
			nbytes = (*myfactGPU)*sizeof(struct start_info);

			// copy memory address from PENELOPE variables
			myimage = (unsigned long long int*)gpumemaddr_.gpuimage;
			num_detected_primary = (int*)gpumemaddr_.gpudetect;
			h_a = (struct start_info*)gpumemaddr_.hosta;
			d_a = (struct start_info*)gpumemaddr_.deva;
			pitch = (size_t)gpumemaddr_.devpitch;
			
			// assign number of threads and blocks
			threads = dim3(BLOCKSIZE,1);
			blocks = dim3(GRIDSIZE,1);

			// reading data from buffer
			for(my_index = 0; my_index < (*myfactGPU); my_index++)		// iterate over buffer length
			{
				// units in the penelope output file are in cm. Convert them to microns.
				h_a[my_index].str_x = optical_.xbufopt[my_index] * 10000.0f;	// x-coordinate of interaction event
				h_a[my_index].str_y = optical_.ybufopt[my_index] * 10000.0f;	// y-coordinate
				h_a[my_index].str_z = optical_.zbufopt[my_index] * 10000.0f;	// z-coordinate
				h_a[my_index].str_E = optical_.debufopt[my_index];		// energy deposited
				h_a[my_index].str_histnum = optical_.nbufopt[my_index];		// x-ray history number
				
				// sample # optical photons based on light yield and energy deposited for this interaction event (using Poisson distribution)
				mu_gsl = (double)h_a[my_index].str_E * yield;
				h_a[my_index].str_N = gsl_ran_poisson(rgsl,mu_gsl);
				
				if(h_a[my_index].str_N > max_photon_per_EDE)
				{
					printf("\n\n str_n exceeds max photons. program is exiting - %d !! \n\n", h_a[my_index].str_N);
					exit(0);
				}
			} 
			
			hipMemset(dPhotonHist, 0, numPhotonHist*sizeof(struct histStruct));
			memset(hPhotonHist, 0, numPhotonHist*sizeof(struct histStruct));
			
			// execute the optical transport kernel 
			// asynchronously copy data from host to device	(all to stream 0)
			cutilSafeCall( hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice, 0) );
			
			if(finishPhotonHist < numPhotonHist)
			{
				boolToCollectData = 1;
			}
			else
			{
				boolToCollectData = 0;
			}
			
			// each kernel has BLOCKSIZE threads; each thread transports one event in the buffer (info.str_N optical photons)
			algo<<<blocks, threads, 0, 0>>>(d_a, myimage, num_detected_primary, pitch, (*myfactGPU), xdetector, ydetector, radius, height, 
			n_C, n_IC, top_absfrac, bulk_abscoeff, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, d_max, sensorRefl, dPhotonHist, boolToCollectData); 

			// asynchronously copy data from device to host
			h_myimage = han_h_myimage;
			cutilSafeCall( hipMemcpy2DAsync((void*)h_myimage,sizeof(unsigned long long int)*xdim,(void*)myimage,pitch, sizeof(unsigned long long int)*xdim,ydim,hipMemcpyDeviceToHost, 0) );
			han_h_myimage = h_myimage;
			
			cutilSafeCall( hipMemcpyAsync(hPhotonHist, dPhotonHist, numPhotonHist*sizeof(struct histStruct), hipMemcpyDeviceToHost) );
			han_h_photonHist = (unsigned long long int *)hPhotonHist;
			han_d_photonHist = (unsigned long long int *)dPhotonHist;
			
			h2_num_detected_primary = vis_num_detected_primary;
			cutilSafeCall( hipMemcpyAsync(h2_num_detected_primary, num_detected_primary, sizeof(int)*num_primary, hipMemcpyDeviceToHost, 0) );
			vis_num_detected_primary = h2_num_detected_primary;
			
			cutilCheckMsg("algo() execution failed\n");
		}	
		else if(*penctr == 101)		// calling optical transport kernel last time. copy back final data from device to host
		{

			// synchronize threads to ensure that GPU is not busy processing previous kernel call
			hipDeviceSynchronize();
			// here 'nbytes' is not necesaarily equal to 'gpubufsize*sizeof(struct start_info)', because in the last call optical_.myctropt can be <= gpubufsize
			nbytes = (*myfactGPU)*sizeof(struct start_info);

			// copy memory address from PENELOPE variables
			myimage = (unsigned long long int*)gpumemaddr_.gpuimage;
			num_detected_primary = (int*)gpumemaddr_.gpudetect;
			h_a = (struct start_info*)gpumemaddr_.hosta;
			d_a = (struct start_info*)gpumemaddr_.deva;
			pitch = (size_t)gpumemaddr_.devpitch;

			// allocate host memory
			//unsigned long long int *h_myimage = 0;          // page-locked host memory for asynchronous copying (contain host image for evey kernel run)
			//allocate memory for h_myimage
			//cutilSafeCall( hipHostMalloc((void**)&h_myimage, xdim*ydim*sizeof(unsigned long long int)) ); 
			h_myimage = han_h_myimage;
			hPhotonHist = (struct histStruct *) han_h_photonHist;
			dPhotonHist = (struct histStruct *) han_d_photonHist;
			
			int *h_num_detected_primary = 0;		// host memory to get # detected/primary
			cutilSafeCall( hipHostMalloc((void**)&h_num_detected_primary, sizeof(int)*num_primary) );

			for(indexj=0; indexj < num_primary; indexj++)
			  h_num_detected_primary[indexj] = 0;
			  
			int *h_histogram = 0;		// host memory for storing histogram of # photons detected/primary
			h_histogram = (int*)malloc(sizeof(int)*num_bins);
			
			for(indexj=0; indexj < num_bins; indexj++)
			  h_histogram[indexj] = 0;

			// assign number of threads and blocks
			threads = dim3(BLOCKSIZE,1);
			blocks = dim3(GRIDSIZE,1);


			// reading data from buffer
			for(my_index = 0; my_index < (*myfactGPU); my_index++)		// iterate over x-rays
			{
				// units in the penelope output file are in cm. Convert them to microns.
				h_a[my_index].str_x = optical_.xbufopt[my_index] * 10000.0f;	// x-coordinate
				h_a[my_index].str_y = optical_.ybufopt[my_index] * 10000.0f;	// y-coordinate
				h_a[my_index].str_z = optical_.zbufopt[my_index] * 10000.0f;	// z-coordinate
				h_a[my_index].str_E = optical_.debufopt[my_index];		// energy deposited
				h_a[my_index].str_histnum = optical_.nbufopt[my_index];		// x-ray history number

				// sample # optical photons based on light yield and energy deposited for this interaction event
				mu_gsl = (double)h_a[my_index].str_E * yield;
				h_a[my_index].str_N = gsl_ran_poisson(rgsl,mu_gsl);

				if(h_a[my_index].str_N > max_photon_per_EDE)
				{
					printf("\n\n str_n exceeds max photons. program is exiting - %d !! \n\n", h_a[my_index].str_N);
					exit(0);
				}

			} // for loop ends

			// execute the kernel 
			// asynchronously copy data from host to device	(all to stream 0)
			cutilSafeCall( hipMemcpyAsync(d_a, h_a, nbytes, hipMemcpyHostToDevice, 0) );

			// each kernel has BLOCKSIZE threads; each thread transports one event in the buffer (info.str_N optical photons)
			algo<<<blocks, threads, 0, 0>>>(d_a, myimage, num_detected_primary, pitch, (*myfactGPU), xdetector, ydetector, radius, height, n_C, n_IC, 
			top_absfrac, bulk_abscoeff, beta, d_min, pixelsize, lbound_x, lbound_y, ubound_x, ubound_y, d_max, sensorRefl, dPhotonHist, 0); 

			// asynchronously copy image data from device to host
			cutilSafeCall( hipMemcpy2DAsync((void*)h_myimage,sizeof(unsigned long long int)*xdim,(void*)myimage,pitch,sizeof(unsigned long long int)*xdim,ydim,hipMemcpyDeviceToHost, 0) );
			cutilSafeCall( hipMemcpyAsync(h_num_detected_primary, num_detected_primary, sizeof(int)*num_primary, hipMemcpyDeviceToHost, 0) );

			cutilCheckMsg("algo() execution failed\n");

			hipDeviceSynchronize();	// ensure that GPU has finished before copying back the final results.

			// copy counters from device to host
			cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_detect,num_detect,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));	
			cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_generated,num_generated,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));	
			cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_abs_top,num_abs_top,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));	
			cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_abs_bulk,num_abs_bulk,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));
			cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_lost,num_lost,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));
			cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_outofcol,num_outofcol,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));
			cutilSafeCall(hipMemcpyFromSymbol((void *) &host_num_theta1,num_theta1,sizeof(unsigned long long int)*1,0,hipMemcpyDeviceToHost));

			// add h_myimage to the new_myimage (array in PENELOPE)
			for(indexi = 0; indexi < ydim; indexi++)
		 	 for(indexj = 0; indexj < xdim; indexj++)
				outputimage_.newimageopt[indexi][indexj] = outputimage_.newimageopt[indexi][indexj] + h_myimage[indexi*xdim + indexj];

			// make histogram of number of detected photons/primary for num_bins
			int binsize=0, newbin=0;
			int bincorr=0;
							
			binsize = floor((max_optphotons-min_optphotons)/num_bins);	// calculate size of each bin. Assuming equally spaced bins.
			bincorr = floor(min_optphotons/binsize);			// correction in bin number if min_optphotons > 0.
			
			for(indexi = 0; indexi < num_primary; indexi++)
			 {
			 	newbin = floor(h_num_detected_primary[indexi]/binsize) - bincorr;	// find bin #
			 	
 			 	if(h_num_detected_primary[indexi] > 0)	// store only non-zero bins
 			 	{
				 	if(h_num_detected_primary[indexi] <= min_optphotons)	// # detected < minimum photons given by user, add to the first bin
						h_histogram[0]++;
				 	else if(h_num_detected_primary[indexi] >= max_optphotons)	// # detected > maximum photons given by user, then add to the last bin
			 			h_histogram[num_bins-1]++;
			 		else
				 		h_histogram[newbin]++; 
				}
			 }
			
			// add num_detected_primary to gldetprimary array in PENELOPE
			for(indexi = 0; indexi < num_bins; indexi++)
				outputdetprim_.gldetprimary[indexi] = outputdetprim_.gldetprimary[indexi] + h_histogram[indexi];

		   
			// type cast unsigned long long int to double
			double cast_host_num_generated;
			double cast_host_num_detect;
			double cast_host_num_abs_top;
			double cast_host_num_abs_bulk;
			double cast_host_num_lost;
			double cast_host_num_outofcol;
			double cast_host_num_theta1;
			double cast_gputime;

			cast_host_num_generated = (double)host_num_generated;
			cast_host_num_detect    = (double)host_num_detect;
			cast_host_num_abs_top   = (double)host_num_abs_top;
			cast_host_num_abs_bulk  = (double)host_num_abs_bulk;
			cast_host_num_lost      = (double)host_num_lost;
			cast_host_num_outofcol  = (double)host_num_outofcol;
			cast_host_num_theta1    = (double)host_num_theta1;
			cast_gputime		= (double)totalgpuTime;

			 // save to global counters
			 optstats_.glgen      = optstats_.glgen      + cast_host_num_generated;
			 optstats_.gldetect   = optstats_.gldetect   + cast_host_num_detect;
			 optstats_.glabstop   = optstats_.glabstop   + cast_host_num_abs_top;
			 optstats_.glabsbulk  = optstats_.glabsbulk  + cast_host_num_abs_bulk;
			 optstats_.gllost     = optstats_.gllost     + cast_host_num_lost;
			 optstats_.gloutofcol = optstats_.gloutofcol + cast_host_num_outofcol;
			 optstats_.gltheta1   = optstats_.gltheta1   + cast_host_num_theta1;
			 optstats_.glgputime  = optstats_.glgputime  + cast_gputime;


	 
			// release resources
			cutilSafeCall(hipFree(d_a));
			cutilSafeCall(hipFree(myimage));
			cutilSafeCall(hipFree(num_detected_primary));
			cutilSafeCall(hipFree(dPhotonHist));
			hipHostFree(h_a);
			hipHostFree(h_myimage);
			hipHostFree(h_num_detected_primary);
			hipHostFree(hPhotonHist);

			free(h_histogram);
		}	// else ends


		return;
	}	// CUDA main() ends
	
#endif


